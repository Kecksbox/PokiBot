#include "hip/hip_runtime.h"
﻿#ifndef __main__
#define __main__

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Trainer.cuh"
#include "GameMaster.cuh"
#include "Logger.cuh"
#include "RaiseBuckets.cuh"

#include <cstdio>
#include <vector>
#include <time.h>
#include <string>

using std::cout;
using std::endl;
using std::cin;
using std::vector;

vector<string> CONSOLE_OPTIONS = { "clear", "train", "play", "benchmark", "exit" };
vector<string> DEVICE_OPTIONS = { "cpu", "gpu" };
vector<string> PLAY_OPTIONS = { "vsRandom" };
string GET_ITERATIONS = "Input number of iterations: ";
string GET_WRONG_INPUT = "Falsche Eingabe ... zurück zur Hauptauswahl";
string FOLDER = "outputs";

void clearFiles(std::string folder, std::string filePrefix) {
    using std::remove;

    remove((folder + "/" + filePrefix + "_buckets_0").c_str());
    remove((folder + "/" + filePrefix + "_buckets_1").c_str());
    remove((folder + "/" + filePrefix + "_buckets_2").c_str());
    remove((folder + "/" + filePrefix + "_buckets_3").c_str());

    remove((folder + "/" + filePrefix + "00").c_str());
    remove((folder + "/" + filePrefix + "00").c_str());

    remove((folder + "/" + filePrefix + "00").c_str());
    remove((folder + "/" + filePrefix + "00").c_str());

    remove((folder + "/" + filePrefix + "00").c_str());
    remove((folder + "/" + filePrefix + "00").c_str());

    remove((folder + "/" + filePrefix + "00").c_str());
    remove((folder + "/" + filePrefix + "00").c_str());
}

void benchmark() {
    clearFiles(FOLDER, "blueprint");
    int trainIterations = 10000;
    int maxIterations = 500000;
    int playIterations = 100000;
    Logger::initBenchmark(FOLDER, "benchmark", DEVICE_OPTIONS.at(1), BLOCKSIZE, raiseSizes, trainIterations, maxIterations, playIterations);
    for (int currentIteration = trainIterations; currentIteration < maxIterations; currentIteration+= trainIterations) {
        std::chrono::system_clock::time_point initStart, trainStart, trainFinish;
        //gpu
        Logger::logStart(DEVICE_OPTIONS.at(1), BLOCKSIZE, trainIterations);

        initStart = std::chrono::system_clock::now();
        TexasHoldemTrainer trainer = TexasHoldemTrainer(FOLDER, "blueprint");
        trainStart = std::chrono::system_clock::now();
        Logger::logInit(initStart, trainStart);

        trainer.trainSequentiell(trainIterations, true);
        trainFinish = std::chrono::system_clock::now();
        Logger::logTraining(trainStart, trainFinish, trainIterations);

        GameMaster gameMaster = GameMaster(FOLDER, "blueprint");
        //PlayResult* result = gameMaster.playBlueprintVersusRandom(playIterations);
        PlayResult* result = gameMaster.playBlueprintVersusBlueprint(playIterations);

        Logger::logPlay(result, playIterations);

        trainer.schablone->roundInfos.at(3).at(0).bucketFunction->loadBucketFunction();
        size_t bucketListSize = trainer.schablone->roundInfos.at(3).at(0).bucketFunction->bucketList.size();
        size_t bucketSize = trainer.schablone->roundInfos.at(3).at(0).bucketFunction->size * 2;
        size_t bucketCount = bucketListSize / bucketSize;

        std::string fileSize = trainer.schablone->roundInfos.at(3).at(0).blueprintHandler->getFileSize();

        Logger::logBenchmark(FOLDER, "benchmark", currentIteration, playIterations, fileSize, bucketCount, initStart, trainStart, trainFinish, result);

        free(result);
    }

    
}

std::string getOptions(std::vector<string> options) {
    std::ostringstream optionsString;
    for (int i = 0; i < options.size(); i++) {
        optionsString << options.at(i) << " (" << i << ")" << (i == options.size()-1 ? ": " : ", ");
    }
    return optionsString.str();
}

void clear() {
   clearFiles(FOLDER, "blueprint");
    cout << "cleared successfully" << endl;
   /* int deviceOption;

    cout << "device: cpu(0), gpu(1), all(2)";
    cin >> deviceOption;
    if (deviceOption == '0') {
        clearFiles("cpu");
    }
    else if (deviceOption == '1') {
        clearFiles("gpu");
    }
    else if (deviceOption == '2') {
        clearFiles("cpu");
        clearFiles("gpu");
    }
    else {
        std::cout << "deletion aborted" << std::endl;
    }*/
}

void play() {
    int playOption, iterations;

    cout << GET_ITERATIONS;
    cin >> iterations;
    GameMaster gameMaster = GameMaster(FOLDER, "blueprint");
    //PlayResult* result = gameMaster.playBlueprintVersusRandom(iterations);
    PlayResult* result = gameMaster.playBlueprintVersusBlueprint(iterations);

    Logger::logPlay(result, iterations);
    
    free(result);
}

void train() {
    int deviceOption, iterations;
    std::chrono::system_clock::time_point initStart, trainStart, trainFinish;

    cout << getOptions(DEVICE_OPTIONS);
    cin >> deviceOption;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    cout << GET_ITERATIONS;
    cin >> iterations;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    Logger::logStart(DEVICE_OPTIONS.at(deviceOption), BLOCKSIZE, iterations);

    initStart = std::chrono::system_clock::now();
    TexasHoldemTrainer trainer = TexasHoldemTrainer(FOLDER,  "blueprint");
    trainStart = std::chrono::system_clock::now();
    Logger::logInit(initStart, trainStart);

    trainer.trainSequentiell(iterations, deviceOption == 1);
    trainFinish = std::chrono::system_clock::now();
    Logger::logTraining(trainStart, trainFinish, iterations);

    if (gDebug) {
        for (int i = 0; i < trainer.elapsedKernelTimes.size(); i++) {
            Logger::logToConsole(std::to_string(trainer.elapsedKernelTimes.at(i) /= iterations) + " ns");
        }
        for (int i = 0; i < trainer.elapsedCpuTimes.size(); i++) {
            Logger::logToConsole(std::to_string(trainer.elapsedCpuTimes.at(i) /= iterations) + " ns");
        }
        for (int i = 0; i < trainer.elapsedMemcpyTimes.size(); i++) {
            Logger::logToConsole(std::to_string(trainer.elapsedMemcpyTimes.at(i) /= iterations) + " ns");
        }
    }

}

int main() {
    srand(0);

    int consoleOption;

    while (true) {
        cout << getOptions(CONSOLE_OPTIONS);
        cin >> consoleOption;

        if (!cin) {
            cout << GET_WRONG_INPUT << endl;
            std::cin.clear();
            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
            continue;
        }

        switch (consoleOption) {
        case 0:
            clear();
            break;
        case 1:
            train();
            break;
        case 2:
            play();
            break;
        case 3:
            benchmark();
            break;
        case 4:
            return;
        }
    }
}
#endif