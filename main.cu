#include "hip/hip_runtime.h"
﻿#ifndef __main__
#define __main__

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Trainer.cuh"
#include "GameMaster.cuh"
#include "Logger.cuh"

#include <cstdio>
#include <vector>
#include <time.h>
#include <string>

using std::cout;
using std::endl;
using std::cin;
using std::vector;

vector<string> CONSOLE_OPTIONS = { "clear", "train", "play", "benchmark", "exit" };
vector<string> DEVICE_OPTIONS = { "cpu", "gpu" };
vector<string> PLAY_OPTIONS = { "vsRandom" };
string GET_ITERATIONS = "Input number of iterations: ";
string GET_WRONG_INPUT = "Falsche Eingabe ... zurück zur Hauptauswahl";

void benchmark() {
    std::chrono::system_clock::time_point initStart, trainStart, trainFinish;
    int iterations = 1000;
    //gpu
    Logger::logStart(DEVICE_OPTIONS.at(1), BLOCKSIZE, iterations);

    initStart = std::chrono::system_clock::now();
    TexasHoldemTrainer trainer = TexasHoldemTrainer("blueprint");
    trainStart = std::chrono::system_clock::now();
    Logger::logInit(initStart, trainStart);

    trainer.trainSequentiell(iterations, true);
    trainFinish = std::chrono::system_clock::now();
    Logger::logTraining(trainStart, trainFinish, 1000);

    GameMaster gameMaster = GameMaster("blueprint");
    PlayResult result = gameMaster.playBlueprintVersusRandom(iterations);

    Logger::logPlay(result.winCounters.at(0), result.winCounters.at(1), result.payoffCounters.at(0), result.payoffCounters.at(1), iterations);

    trainer.schablone->roundInfos.at(3).at(0).bucketFunction->loadBucketFunction();
    size_t bucketListSize = trainer.schablone->roundInfos.at(3).at(0).bucketFunction->bucketList.size();
    size_t bucketSize = trainer.schablone->roundInfos.at(3).at(0).bucketFunction->size * 2;
    size_t bucketCount = bucketListSize / bucketSize;

    Logger::logBenchmark(1, iterations, bucketCount, initStart, trainStart, trainFinish, result.winCounters.at(0), result.payoffCounters.at(0));
}

void clearFiles() {
    using std::remove;

    remove("blueprint_buckets_0");
    remove("blueprint_buckets_1");
    remove("blueprint_buckets_2");
    remove("blueprint_buckets_3");

    remove("blueprint00");
    remove("blueprint01");
    remove("blueprint10");
    remove("blueprint11");
    remove("blueprint20");
    remove("blueprint21");
    remove("blueprint30");
    remove("blueprint31");
}

std::string getOptions(std::vector<string> options) {
    std::ostringstream optionsString;
    for (int i = 0; i < options.size(); i++) {
        optionsString << options.at(i) << " (" << i << ")" << (i == options.size()-1 ? ": " : ", ");
    }
    return optionsString.str();
}

void clear() {
    clearFiles();
    cout << "cleared successfully" << endl;
   /* int deviceOption;

    cout << "device: cpu(0), gpu(1), all(2)";
    cin >> deviceOption;
    if (deviceOption == '0') {
        clearFiles("cpu");
    }
    else if (deviceOption == '1') {
        clearFiles("gpu");
    }
    else if (deviceOption == '2') {
        clearFiles("cpu");
        clearFiles("gpu");
    }
    else {
        std::cout << "deletion aborted" << std::endl;
    }*/
}

void play() {
    int playOption, iterations;

    cout << GET_ITERATIONS;
    cin >> iterations;
    GameMaster gameMaster = GameMaster("blueprint");
    PlayResult result = gameMaster.playBlueprintVersusRandom(iterations);

    Logger::logPlay(result.winCounters.at(0), result.winCounters.at(1), result.payoffCounters.at(0), result.payoffCounters.at(1), iterations);
}

void train() {
    int deviceOption, iterations;
    std::chrono::system_clock::time_point initStart, trainStart, trainFinish;

    cout << getOptions(DEVICE_OPTIONS);
    cin >> deviceOption;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    cout << GET_ITERATIONS;
    cin >> iterations;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    Logger::logStart(DEVICE_OPTIONS.at(deviceOption), BLOCKSIZE, iterations);

    initStart = std::chrono::system_clock::now();
    TexasHoldemTrainer trainer = TexasHoldemTrainer("blueprint");
    trainStart = std::chrono::system_clock::now();
    Logger::logInit(initStart, trainStart);

    trainer.trainSequentiell(iterations, deviceOption == 1);
    trainFinish = std::chrono::system_clock::now();
    Logger::logTraining(trainStart, trainFinish, iterations);

    if (gDebug) {
        for (int i = 0; i < 3; i++) {
            Logger::logToConsole(std::to_string(trainer.elapsedKernelTimes.at(i) /= iterations));
        }
    }

}

int main() {
    srand(0);

    int consoleOption;

    while (true) {
        cout << getOptions(CONSOLE_OPTIONS);
        cin >> consoleOption;

        if (!cin) {
            cout << GET_WRONG_INPUT << endl;
            std::cin.clear();
            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
            continue;
        }

        switch (consoleOption) {
        case 0:
            clear();
            break;
        case 1:
            train();
            break;
        case 2:
            play();
            break;
        case 3:
            benchmark();
            break;
        case 4:
            return;
        }
    }
}
#endif