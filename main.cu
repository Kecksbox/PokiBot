#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Trainer.cuh"
#include "GameMaster.cuh"

#include <cstdio>
#include <vector>
#include <time.h>
#include <string>

void clearFiles(std::string prefix) {
    std::remove((prefix + "_blueprint_buckets_0").c_str());
    std::remove((prefix + "_blueprint_buckets_1").c_str());
    std::remove((prefix + "_blueprint_buckets_2").c_str());
    std::remove((prefix + "_blueprint_buckets_3").c_str());

    std::remove((prefix + "_blueprint00").c_str());
    std::remove((prefix + "_blueprint01").c_str());
    std::remove((prefix + "_blueprint10").c_str());
    std::remove((prefix + "_blueprint11").c_str());
    std::remove((prefix + "_blueprint20").c_str());
    std::remove((prefix + "_blueprint21").c_str());
    std::remove((prefix + "_blueprint30").c_str());
    std::remove((prefix + "_blueprint31").c_str());
}

int main() {
    srand(0);

    bool running = true;
    char action;
    int iterations;
    char device;
    clock_t init, train;

    while (running) {
        std::cout << "What do you want to do? c(learfiles), t(rain), p(lay), e(xit) ";
        std::cin >> action;

        if (action == 'e') {
            running = false;
        }
        else if (action == 'c') {
            std::cout << "device: cpu(0), gpu(1), all(2)";
            std::cin >> device;
            if (device == '0') {
                clearFiles("cpu");
            }
            else if (device == '1') {
                clearFiles("gpu");
            }
            else if (device == '2') {
                clearFiles("cpu");
                clearFiles("gpu");
            }
            else {
                std::cout << "deletion aborted" << std::endl;
            }
        }
        else if (action == 't') {
            std::cout << "device: (c)pu, (g)pu: ";
            std::cin >> device;
            if (device == 'c') {
                std::cout << "Input number of iterations: ";
                std::cin >> iterations;
                init = clock();
                TexasHoldemTrainer trainer = TexasHoldemTrainer("blueprint");
                init = clock() - init;
                trainer.trainSequentiell(iterations, false);
                train = clock() - init;
                std::cout << "init: " << init << " train " << iterations << " iterations: " << train << " (" << train / static_cast<double>(iterations) << " pro iteration)" << std::endl;
            }
            else {
                std::cout << "Input number of iterations: ";
                std::cin >> iterations;
                init = clock();
                TexasHoldemTrainer trainer = TexasHoldemTrainer("blueprint");
                init = clock() - init;
                trainer.trainSequentiell(iterations, true);
                train = clock() - init;
                std::cout << "init: " << init << " train " << iterations << " iterations: " << train << " (" << train / static_cast<double>(iterations) << " pro iteration)" << std::endl;
            }
            
        } else if (action == 'p') {
            std::cout << "Input number of iterations: ";
            std::cin >> iterations;
            GameMaster gameMaster = GameMaster("blueprint");
            gameMaster.playBlueprintVersusRandom(iterations);
        } else {
            std::cout << "Ungültige Eingaben";
        }
    }
}
