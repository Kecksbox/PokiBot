#include "hip/hip_runtime.h"
﻿#ifndef __main__
#define __main__

#include "hip/hip_runtime.h"
#include ""

#include "Trainer.cuh"
#include "GameMaster.cuh"
#include "Logger.cuh"
#include "RaiseBuckets.cuh"

#include <stdio.h>
#include <cstdio>
#include <vector>
#include <time.h>
#include <string>
#include <filesystem>

using std::cout;
using std::endl;
using std::cin;
using std::vector;

vector<string> CONSOLE_OPTIONS = { "clear", "train", "play", "benchmark", "exit" };
vector<string> DEVICE_OPTIONS = { "cpu", "gpu" };
vector<string> PLAY_OPTIONS = { "vsRandom" };
string GET_FILENAME = "Input filename (blueprint): ";
string GET_COMPARISON_FILENAME = "Input filename (play vs random): ";
string GET_ITERATIONS = "Input number of iterations: ";
string GET_WRONG_INPUT = "Falsche Eingabe ... zurück zur Hauptauswahl";
string GET_BENCHMARKING_INPUT = "Input filename (blueprint), comparison_1 (comparison), comparison_2 (random), trainMaxIterations (200k), trainIterationSteps (5k), playIterations (25k)";
string FOLDER = "outputs";
string COMPARISON_1 = "comparison";
string COMPARISON_2 = "comparison_2";

void clearFiles(std::string folder, std::string filePrefix) {
    using std::remove;

    remove((folder + "/" + filePrefix + "_buckets_0").c_str());
    remove((folder + "/" + filePrefix + "_buckets_1").c_str());
    remove((folder + "/" + filePrefix + "_buckets_2").c_str());
    remove((folder + "/" + filePrefix + "_buckets_3").c_str());

    remove((folder + "/" + filePrefix + "00").c_str());
    remove((folder + "/" + filePrefix + "00").c_str());

    remove((folder + "/" + filePrefix + "00").c_str());
    remove((folder + "/" + filePrefix + "00").c_str());

    remove((folder + "/" + filePrefix + "00").c_str());
    remove((folder + "/" + filePrefix + "00").c_str());

    remove((folder + "/" + filePrefix + "00").c_str());
    remove((folder + "/" + filePrefix + "00").c_str());
}

void benchmark() {
    string filename, comparison1, comparison2;
    int trainMaxIterations, tranIterationSteps, playIterations;
    bool comparison2Random;
    
    cout << GET_BENCHMARKING_INPUT;

    cin >> filename;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    if (filename == "0") filename = "blueprint";

    cin >> comparison1;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    if (comparison1 == "0") comparison1 = COMPARISON_1;

    cin >> comparison2;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    if (comparison2 == "0") comparison2Random = true;

    cin >> trainMaxIterations;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    if (trainMaxIterations == 0) trainMaxIterations = 200000;

    cin >> tranIterationSteps;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    if (tranIterationSteps == 0) tranIterationSteps = 5000;

    cin >> playIterations;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    if (playIterations == 0) playIterations = 25000;

    Logger::initBenchmark(FOLDER, "benchmark" + comparison1, DEVICE_OPTIONS.at(1), BLOCKSIZE, raiseSizes, tranIterationSteps, trainMaxIterations, playIterations);
    Logger::initBenchmark(FOLDER, "benchmark" + comparison2, DEVICE_OPTIONS.at(1), BLOCKSIZE, raiseSizes, tranIterationSteps, trainMaxIterations, playIterations);

    TexasHoldemTrainer trainer = TexasHoldemTrainer(FOLDER, filename);
    GameMaster gameMaster = GameMaster(FOLDER, filename);

    for (int currentIteration = tranIterationSteps; currentIteration < trainMaxIterations; currentIteration+= tranIterationSteps) {
        std::chrono::system_clock::time_point initStart, trainStart, trainFinish;
        //gpu
        Logger::logStart(DEVICE_OPTIONS.at(1), BLOCKSIZE, tranIterationSteps);

        initStart = std::chrono::system_clock::now();
        
        trainStart = std::chrono::system_clock::now();
        Logger::logInit(initStart, trainStart);

        trainer.trainSequentiell(tranIterationSteps, true);
        trainFinish = std::chrono::system_clock::now();
        Logger::logTraining(trainStart, trainFinish, tranIterationSteps);

        
        PlayResult* result = gameMaster.playBlueprintVersusBlueprint(playIterations, comparison1);
        PlayResult* result2;

        if (comparison2Random) {
            result2 = gameMaster.playBlueprintVersusRandom(playIterations);
        }
        else {
            result2 = gameMaster.playBlueprintVersusBlueprint(playIterations, comparison2);
        }

        Logger::logPlay(result, playIterations);
        Logger::logPlay(result2, playIterations);
        

        trainer.schablone->roundInfos.at(3).at(0).bucketFunction->loadBucketFunction();
        size_t bucketListSize = trainer.schablone->roundInfos.at(3).at(0).bucketFunction->bucketList.size();
        size_t bucketSize = trainer.schablone->roundInfos.at(3).at(0).bucketFunction->size * 2;
        size_t bucketCount = bucketListSize / bucketSize;

        std::string fileSize = trainer.schablone->roundInfos.at(3).at(0).blueprintHandler->getFileSize();

        Logger::logBenchmark(FOLDER, "benchmark" + comparison1, currentIteration, playIterations, fileSize, bucketCount, initStart, trainStart, trainFinish, result);

        Logger::logBenchmark(FOLDER, "benchmark" + comparison2, currentIteration, playIterations, fileSize, bucketCount, initStart, trainStart, trainFinish, result2);


        free(result);
    }

    
}

std::string getOptions(std::vector<string> options) {
    std::ostringstream optionsString;
    for (int i = 0; i < options.size(); i++) {
        optionsString << options.at(i) << " (" << i << ")" << (i == options.size()-1 ? ": " : ", ");
    }
    return optionsString.str();
}

void clear() {
   clearFiles(FOLDER, "blueprint");
    cout << "cleared successfully" << endl;
   /* int deviceOption;

    cout << "device: cpu(0), gpu(1), all(2)";
    cin >> deviceOption;
    if (deviceOption == '0') {
        clearFiles("cpu");
    }
    else if (deviceOption == '1') {
        clearFiles("gpu");
    }
    else if (deviceOption == '2') {
        clearFiles("cpu");
        clearFiles("gpu");
    }
    else {
        std::cout << "deletion aborted" << std::endl;
    }*/
}

void play() {
    int playOption, iterations;
    string filename, comparisonFilename;

    cout << GET_ITERATIONS;
    cin >> iterations;
    
    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    cout << GET_FILENAME;
    cin >> filename;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    if (filename == "0") filename = "blueprint";

    cout << GET_COMPARISON_FILENAME;
    cin >> comparisonFilename;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }


    GameMaster gameMaster = GameMaster(FOLDER, filename);
    PlayResult* result;

    if (comparisonFilename == "0") {
        gameMaster.playBlueprintVersusRandom(iterations);
    }
    else {
        gameMaster.playBlueprintVersusBlueprint(iterations, comparisonFilename);
    }

    Logger::logPlay(result, iterations);
    
    free(result);
}

void train() {
    int deviceOption, iterations;
    string filename;
    std::chrono::system_clock::time_point initStart, trainStart, trainFinish;

    cout << getOptions(DEVICE_OPTIONS);
    cin >> deviceOption;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    cout << GET_ITERATIONS;
    cin >> iterations;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    cout << GET_FILENAME;
    cin >> filename;

    if (!cin) {
        cout << GET_WRONG_INPUT << endl;
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        return;
    }

    if (filename == "0") filename = "blueprint";

    Logger::logStart(DEVICE_OPTIONS.at(deviceOption), BLOCKSIZE, iterations);

    initStart = std::chrono::system_clock::now();
    TexasHoldemTrainer trainer = TexasHoldemTrainer(FOLDER,  filename);
    trainStart = std::chrono::system_clock::now();
    Logger::logInit(initStart, trainStart);

    trainer.trainSequentiell(iterations, deviceOption == 1);
    trainFinish = std::chrono::system_clock::now();
    Logger::logTraining(trainStart, trainFinish, iterations);

    if (gDebug) {
        for (int i = 0; i < trainer.elapsedKernelTimes.size(); i++) {
            Logger::logToConsole(std::to_string(trainer.elapsedKernelTimes.at(i) /= iterations) + " ns");
        }
        for (int i = 0; i < trainer.elapsedCpuTimes.size(); i++) {
            Logger::logToConsole(std::to_string(trainer.elapsedCpuTimes.at(i) /= iterations) + " ns");
        }
        for (int i = 0; i < trainer.elapsedMemcpyTimes.size(); i++) {
            Logger::logToConsole(std::to_string(trainer.elapsedMemcpyTimes.at(i) /= iterations) + " ns");
        }
    }

}

int main() {
    //srand(0);
    srand(std::chrono::system_clock::now().time_since_epoch().count());

    int consoleOption;

    while (true) {
        cout << getOptions(CONSOLE_OPTIONS);
        cin >> consoleOption;

        if (!cin) {
            cout << GET_WRONG_INPUT << endl;
            std::cin.clear();
            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
            continue;
        }

        switch (consoleOption) {
        case 0:
            clear();
            break;
        case 1:
            train();
            break;
        case 2:
            play();
            break;
        case 3:
            benchmark();
            break;
        case 4:
            return;
        }
    }
}
#endif