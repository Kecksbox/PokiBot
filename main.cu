#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Trainer.cuh"

#include <cstdio>
#include <vector>

void clearFiles(std::string prefix) {
    std::remove((std::string() + prefix + "_buckets_0").c_str());
    std::remove((std::string() + prefix + "_buckets_1").c_str());
    std::remove((std::string() + prefix + "_buckets_2").c_str());
    std::remove((std::string() + prefix + "_buckets_3").c_str());

    std::remove((std::string() + prefix + "_00").c_str());
    std::remove((std::string() + prefix + "_01").c_str());
    std::remove((std::string() + prefix + "_10").c_str());
    std::remove((std::string() + prefix + "_11").c_str());
    std::remove((std::string() + prefix + "_20").c_str());
    std::remove((std::string() + prefix + "_21").c_str());
    std::remove((std::string() + prefix + "_30").c_str());
    std::remove((std::string() + prefix + "_31").c_str());
}

int main() {
    srand(0);

    bool running = true;
    char action;
    int iterations;
    int mode;

    while (running) {
        std::cout << "What do you want to do? b(enchmark), c(learfiles), t(rain), p(lay), e(xit) ";
        std::cin >> action;

        if (action == 'e') {
            running = false;
        }
        else if (action == 'c') {
            // clearFiles();
        }
        else if (action == 't') {
            std::cout << "Input number of iterations";
            std::cin >> iterations;
            TexasHoldemTrainer trainer = TexasHoldemTrainer("blueprint");
            trainer.trainGpu(iterations);
        }
        else if (action == 'b') {
            std::cout << "Input number of iterations";
            std::cin >> iterations;
            clock_t t1, t2;
            clearFiles("benchmark");
            {
            TexasHoldemTrainer trainer = TexasHoldemTrainer("benchmark");
            t1 = clock();
            trainer.trainGpu(iterations);
            t1 = clock() - t1;
            clearFiles("benchmark");
            }
            {
            TexasHoldemTrainer trainer = TexasHoldemTrainer("benchmark");
            t2 = clock();
            trainer.trainSequentiell(iterations);
            }
            t2 = clock() - t2;
            std::cout << "gpu: " << t1 << " sequentiell: " << t2 << std::endl;
        } else {
            std::cout << "Ungültige Eingaben";
        }
    }
}
