﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Trainer.cuh"

#include <cstdio>
#include <vector>

void clearFiles() {
    std::remove("blueprint_buckets_0");
    std::remove("blueprint_buckets_1");
    std::remove("blueprint_buckets_2");
    std::remove("blueprint_buckets_3");

    std::remove("blueprint00");
    std::remove("blueprint01");
    std::remove("blueprint10");
    std::remove("blueprint11");
    std::remove("blueprint20");
    std::remove("blueprint21");
    std::remove("blueprint30");
    std::remove("blueprint31");
}

int main() {
    srand(0);

    bool running = true;
    char action;
    int iterations;
    int mode;

    while (running) {
        std::cout << "What do you want to do? c(learfiles), t(rain), p(lay), e(xit) ";
        std::cin >> action;

        if (action == 'e') {
            running = false;
        }
        else if (action == 'c') {
            // clearFiles();
        }
        else if (action == 't') {
            std::cout << "Input number of iterations";
            std::cin >> iterations;
            TexasHoldemTrainer trainer = TexasHoldemTrainer("blueprint");
            trainer.trainSequentiell(iterations);
        } else {
            std::cout << "Ungültige Eingaben";
        }
    }
}
