#include "hip/hip_runtime.h"
#include "RandomAkteur.cuh"
#include "Utils.cuh"
#include "RaiseBuckets.cuh"

#include <random>
#include <algorithm>
#include <iostream>
#include <cstring>

RandomAkteur::~RandomAkteur() {
    delete schablone;
}

RandomAkteur::RandomAkteur(std::string path) {
    schablone = Template::createDefaultTemplate(path);
}

std::pair<char, float> RandomAkteur::act(InformationSet* informationSet) {
    // std::cout << "random akteur action" << std::endl;
    RoundPlayerInfo roundInfo = schablone->roundInfos.at(informationSet->round).at(informationSet->player);
    BucketFunction* bucketFunction = roundInfo.bucketFunction;
    vector<char> bucket = bucketFunction->getBucket(informationSet->playerCardsVisible);
    int bucketPosition = bucketFunction->getBucketPosition(bucket);

    int max = (bucketFunction->bucketList.size()) / (bucketFunction->size * 2);
    bool newBucket = bucketPosition >= max;
    int size = roundInfo.elementSize;

    if (newBucket) {
        return std::pair<char, float>('f', 0.f);
    }
    else {

        std::pair<int, GameState*> result = getCurrentNode(schablone, informationSet->actionHistory);
        int currentNodeWorklistIndex = result.first;
        GameState* currentGameState = result.second;

        if (currentNodeWorklistIndex < schablone->structureList->numStateNodes) {

            std::vector<std::pair<char, float>> currentActions = currentGameState->getActions();

            int actionInt = rand() % currentActions.size();

            for (int i = 0; i < currentActions.size(); i++) {
                if (i == actionInt) {
                    // std::cout << "CHOSEN: " << currentActions.at(i).first << currentActions.at(i).second << std::endl;
                }
                else {
                    // std::cout << currentActions.at(i).first << currentActions.at(i).second << std::endl;
                }

            }
            delete currentGameState;

            return currentActions.at(actionInt);

        }
        else {
            // TODO
            // hier ist eigentlich schon spielende
        }

    }

    return std::pair<char, float>('O', 0.f);
}