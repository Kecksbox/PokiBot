#include "hip/hip_runtime.h"
#include "RandomAkteur.cuh"
#include "Utils.cuh"
#include "RaiseBuckets.cuh"

#include <random>
#include <algorithm>
#include <iostream>
#include <cstring>

RandomAkteur::~RandomAkteur() {
    delete schablone;
}

RandomAkteur::RandomAkteur(std::string folder, std::string fileName) {
    schablone = Template::createDefaultTemplate(folder, fileName);
}

std::pair<char, float> RandomAkteur::act(InformationSet* informationSet) {
    RoundPlayerInfo roundInfo = schablone->roundInfos.at(informationSet->round).at(informationSet->player);
    BucketFunction* bucketFunction = roundInfo.bucketFunction;
    vector<char> bucket = bucketFunction->getBucket(informationSet->playerCardsVisible);
  
        std::pair<int, GameState*> result = getCurrentNode(schablone, informationSet->actionHistory);

        if (result.first < schablone->structureList->numStateNodes) {

            std::vector<std::pair<char, float>> currentActions = result.second->getActions();

            int actionInt = rand() % currentActions.size();

            delete result.second;

            return currentActions.at(actionInt);

        }
        else {
            // TODO
            // hier ist eigentlich schon spielende
        }
        delete result.second;
    

    return std::pair<char, float>('O', 0.f);
}