#include "hip/hip_runtime.h"
#include "RandomAkteur.cuh"
#include "Utils.cuh"
#include "RaiseBuckets.cuh"

#include <random>
#include <algorithm>
#include <iostream>
#include <cstring>

RandomAkteur::~RandomAkteur() {
    delete schablone;
}

RandomAkteur::RandomAkteur(std::string folder, std::string fileName) {
    schablone = Template::createDefaultTemplate(folder, fileName);
}

std::pair<char, float> RandomAkteur::act(InformationSet* informationSet) {
    RoundPlayerInfo roundInfo = schablone->roundInfos.at(informationSet->round).at(informationSet->player);
    BucketFunction* bucketFunction = roundInfo.bucketFunction;
    vector<char> bucket = bucketFunction->getBucket(informationSet->playerCardsVisible);
    int bucketPosition = bucketFunction->getBucketPosition(bucket);

    int max = (bucketFunction->bucketList.size()) / (bucketFunction->size * 2);
    bool newBucket = bucketPosition >= max;

    if (newBucket) {
        return std::pair<char, float>('f', 0.f);
    }
    else {

        std::pair<int, GameState*> result = getCurrentNode(schablone, informationSet->actionHistory);

        if (result.first < schablone->structureList->numStateNodes) {

            std::vector<std::pair<char, float>> currentActions = result.second->getActions();

            int actionInt = rand() % currentActions.size();

            delete result.second;

            return currentActions.at(actionInt);

        }
        else {
            // TODO
            // hier ist eigentlich schon spielende
        }
        delete result.second;
    }

    return std::pair<char, float>('O', 0.f);
}