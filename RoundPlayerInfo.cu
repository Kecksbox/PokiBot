#include "hip/hip_runtime.h"
#include "RoundPlayerInfo.cuh"

#include<iostream>

RoundPlayerInfo::RoundPlayerInfo(int startPointTemplate, int elementSize, BucketFunction* bucketFunction, int round, int player) {
    this->startPointTemplate = startPointTemplate;
    this->elementSize = elementSize;
    this->bucketFunction = bucketFunction;

    this->blueprintHandler = new BlueprintHandler(round, player);
}