#include "hip/hip_runtime.h"
#include "RoundPlayerInfo.cuh"

#include<iostream>

RoundPlayerInfo::RoundPlayerInfo(std::string folder, std::string fileName, int startPointTemplate, int elementSize, BucketFunction* bucketFunction, int round, int player) {
    this->startPointTemplate = startPointTemplate;
    this->elementSize = elementSize;
    this->bucketFunction = bucketFunction;

    this->blueprintHandler = new BlueprintHandler(folder, fileName, round, player);
}