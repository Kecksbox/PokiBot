#include "hip/hip_runtime.h"
#include "RoundPlayerInfo.cuh"

#include<iostream>

RoundPlayerInfo::RoundPlayerInfo(std::string folder, std::string fileName, int startPointTemplate, int elementSize, BucketFunction* bucketFunction, int round, int player) {
    this->startPointTemplate = startPointTemplate;
    this->elementSize = elementSize;
    this->bucketFunction = bucketFunction;

    size_t bucketListSize = bucketFunction->bucketList.size();
    size_t bucketSize = bucketFunction->size * 2;
    size_t bucketCount = bucketListSize / bucketSize;

    this->blueprintHandler = new BlueprintHandler(folder, fileName, round, player, bucketCount);
}