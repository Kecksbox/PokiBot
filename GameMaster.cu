#include "hip/hip_runtime.h"
#include "GameMaster.cuh"
#include "Utils.cuh"
#include "SolverA.cuh"


GameMaster::GameMaster(std::string folder, std::string fileName) {
    this->folder = folder;
    this->fileName = fileName;
}

PlayResult* GameMaster::playBlueprintVersusManual() {
    return nullptr;
}

PlayResult* GameMaster::playBlueprintVersusRandom(int iterations) {
    Template* schablone = Template::createDefaultTemplate(folder, fileName);

    BlueprintAkteur* blueprintAkteur = new BlueprintAkteur(folder, fileName);
    RandomAkteur* randomAkteur = new RandomAkteur(folder, fileName);

    vector<Akteur*> akteure = { blueprintAkteur, randomAkteur };
    vector<Akteur*> rematchAkteure = { randomAkteur, blueprintAkteur };


    PlayResult* playResult = new PlayResult();

    Logger::logToConsole("Play start");
    for (int i = 0; i < iterations; i++) {
        if (i % 10000 == 0) Logger::logIteration(i);

        vector<std::string> cards = getCards();

        std::pair<int, float> result = play(schablone, cards, akteure);
        int winner = result.first;
        float payoff = result.second;
        std::pair<int, float> rematchResult = play(schablone, cards, rematchAkteure);
        int rematchWinner = rematchResult.first;
        float rematchPayoff = rematchResult.second;

        
        if (winner < 0) {
            continue;
        }
        else {
            playResult->winCounters.at(winner)++;
            playResult->payoffCounters.at(winner) += payoff;
            playResult->payoffCounters.at((winner + 1) % 2) -= payoff;
        }

        if (rematchWinner < 0) {
            continue;
        }
        else {
            playResult->rematchWinCounters.at(rematchWinner)++;
            playResult->rematchPayoffCounters.at(rematchWinner) += rematchPayoff;
            playResult->rematchPayoffCounters.at((rematchWinner + 1) % 2) -= rematchPayoff;
        }
    }
    
    delete blueprintAkteur;
    delete randomAkteur;
    delete schablone;

    return playResult;
}

std::pair<int, float> GameMaster::play(Template* schablone, vector<std::string> cards, vector<Akteur*> akteure) {

    InformationSet* informationSet = new InformationSet();
    informationSet->actionHistory = vector<pair<char, float>>();
    informationSet->currentRoundActionHistory = vector<pair<char, float>>();
    informationSet->player = 0;
    informationSet->round = 0;
    informationSet->playerCardsVisible = mapCardsToVisibility(cards, 0, 0);

    while (true) {
        pair<char, float> action = akteure.at(informationSet->player)->act(informationSet);

        if (action.first == 'f') {
            int winner = (informationSet->player + 1) % 2;
            std::pair<int, GameState*> result = getCurrentNode(schablone, informationSet->actionHistory);
            float payoff = winner == 0 ? result.second->pot.second : result.second->pot.first;
            delete result.second;
            delete informationSet;
            return std::pair<int, float>(winner, payoff);
        }
        else if (roundEnd(informationSet->currentRoundActionHistory, action)) {
            if (informationSet->round == 3) {
                vector<vector<string>> playerCards = { {cards.at(0), cards.at(1), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8)}, {cards.at(2), cards.at(3), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8)} };
                int player0Eval = test7(playerCards.at(0));
                int player1Eval = test7(playerCards.at(1));

                bool draw = player0Eval == player1Eval;
                bool playerWon = player0Eval > player1Eval;

                int winner = draw ? -1 : playerWon ? 0 : 1;
                std::pair<int, GameState*> result = getCurrentNode(schablone, informationSet->actionHistory);
                float payoff = 0.f;
                if (!draw) {
                    payoff = winner == 0 ? result.second->pot.second : result.second->pot.first;    
                }

                delete informationSet;
                delete result.second;
                return std::pair<int, float>(winner, payoff);
            }
            else {
                informationSet->round++;
                informationSet->player = 0;
                informationSet->actionHistory.push_back(action);
                informationSet->currentRoundActionHistory = vector<pair<char, float>>();
            }
        }
        else {
            informationSet->player = (informationSet->player + 1) % 2;
            informationSet->actionHistory.push_back(action);
            informationSet->currentRoundActionHistory.push_back(action);
        }
        informationSet->playerCardsVisible = mapCardsToVisibility(cards, informationSet->player, informationSet->round);
    }

    delete informationSet;
}

PlayResult* GameMaster::playBlueprintVersusBlueprint(int iterations, string comparisonBlueprintName) {
    Template* schablone = Template::createDefaultTemplate(folder, fileName);

    BlueprintAkteur* blueprintAkteur = new BlueprintAkteur(folder, fileName);
    BlueprintAkteur* comparisonBlueprintAkteur = new BlueprintAkteur(folder, comparisonBlueprintName + "_" + fileName);

    vector<Akteur*> akteure = { blueprintAkteur, comparisonBlueprintAkteur };
    vector<Akteur*> rematchAkteure = { comparisonBlueprintAkteur, blueprintAkteur };

    PlayResult* playResult = new PlayResult();

    for (int i = 0; i < iterations; i++) {
        //if (i % 1000 == 0) Logger::logIteration(i);

        vector<std::string> cards = getCards();

        std::pair<int, float> result = play(schablone, cards, akteure);
        std::pair<int, float> rematchResult = play(schablone, cards, rematchAkteure);
        int winner = result.first;
        float payoff = result.second;
        int rematchWinner = rematchResult.first;
        float rematchPayoff = rematchResult.second;

        if (winner < 0) {
            continue;
        }
        else {
            playResult->winCounters.at(winner)++;
            playResult->payoffCounters.at(winner) += payoff;
            playResult->payoffCounters.at((winner + 1) % 2) -= payoff;
        }

        if (rematchWinner < 0) {
            continue;
        }
        else {
            playResult->rematchWinCounters.at(rematchWinner)++;
            playResult->rematchPayoffCounters.at(rematchWinner) += rematchPayoff;
            playResult->rematchPayoffCounters.at((rematchWinner + 1) % 2) -= rematchPayoff;
        }
    }

    delete blueprintAkteur;
    delete comparisonBlueprintAkteur;
    delete schablone;

    return playResult;
}