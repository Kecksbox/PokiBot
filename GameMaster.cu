#include "hip/hip_runtime.h"
#include "GameMaster.cuh"
#include "Utils.cuh"
#include "SolverA.cuh"


GameMaster::GameMaster(std::string path) {
    this->path = path;
}

PlayResult GameMaster::playBlueprintVersusRandom(int iterations) {
    Template* schablone = Template::createDefaultTemplate(path);

    BlueprintAkteur* blueprintAkteur = new BlueprintAkteur(path);
    RandomAkteur* randomAkteur = new RandomAkteur(path);
    vector<Akteur*> akteure = { blueprintAkteur, randomAkteur };

    PlayResult playResult = PlayResult();

    
    for (int i = 0; i < iterations; i++) {
        //if (i % 1000 == 0) Logger::logIteration(i);

        vector<std::string> cards = getCards();

        std::pair<int, float> result = play(schablone, cards, akteure);
        int winner = result.first;
        float payoff = result.second;
        
        if (winner < 0) {
            continue;
        }
        else {
            playResult.winCounters.at(winner)++;
            playResult.payoffCounters.at(winner) += payoff;
            playResult.payoffCounters.at((winner + 1) % 2) -= payoff;
        }
    }
    
    delete blueprintAkteur;
    delete randomAkteur;
    delete schablone;

    return playResult;
}

std::pair<int, float> GameMaster::play(Template* schablone, vector<std::string> cards, vector<Akteur*> akteure) {

    InformationSet* informationSet = new InformationSet();
    informationSet->actionHistory = vector<pair<char, float>>();
    informationSet->currentRoundActionHistory = vector<pair<char, float>>();
    informationSet->player = 0;
    informationSet->round = 0;
    informationSet->playerCardsVisible = mapCardsToVisibility(cards, 0, 0);

    while (true) {
        pair<char, float> action = akteure.at(informationSet->player)->act(informationSet);

        if (action.first == 'f') {
            int winner = (informationSet->player + 1) % 2;
            std::pair<int, GameState*> result = getCurrentNode(schablone, informationSet->actionHistory);
            GameState* currentGameState = result.second;
            float payoff = winner == 0 ? currentGameState->pot.second : currentGameState->pot.first;
            delete currentGameState;
            return std::pair<int, float>(winner, payoff);
        }
        else if (roundEnd(informationSet->currentRoundActionHistory, action)) {
            if (informationSet->round == 3) {
                vector<vector<string>> playerCards = { {cards.at(0), cards.at(1), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8)}, {cards.at(2), cards.at(3), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8)} };
                int player0Eval = test7(playerCards.at(0));
                int player1Eval = test7(playerCards.at(1));

                bool draw = player0Eval == player1Eval;
                bool playerWon = player0Eval > player1Eval;

                int winner = draw ? -1 : playerWon ? 0 : 1;
                std::pair<int, GameState*> result = getCurrentNode(schablone, informationSet->actionHistory);
                GameState* currentGameState = result.second;
                float payoff = 0.f;
                if (!draw) {
                    payoff = winner == 0 ? currentGameState->pot.second : currentGameState->pot.first;
                    delete currentGameState;
                }

                return std::pair<int, float>(winner, payoff);
            }
            else {
                informationSet->round++;
                informationSet->player = 0;
                informationSet->actionHistory.push_back(action);
                informationSet->currentRoundActionHistory = vector<pair<char, float>>();
            }
        }
        else {
            informationSet->player = (informationSet->player + 1) % 2;
            informationSet->actionHistory.push_back(action);
            informationSet->currentRoundActionHistory.push_back(action);
        }
        informationSet->playerCardsVisible = mapCardsToVisibility(cards, informationSet->player, informationSet->round);
    }

    delete informationSet;
}

void GameMaster::playBlueprintVersusBlueprint(int iterations) {
    Template* schablone = Template::createDefaultTemplate(path);

    BlueprintAkteur* blueprintAkteur1 = new BlueprintAkteur(path);
    BlueprintAkteur* blueprintAkteur2 = new BlueprintAkteur(path);
    vector<Akteur*> akteure = { blueprintAkteur1, blueprintAkteur2 };

    vector<std::string> cards = getCards();

    play(schablone, cards, akteure);

    delete blueprintAkteur1;
    delete blueprintAkteur2;
    delete schablone;
}