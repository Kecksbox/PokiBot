#include "hip/hip_runtime.h"
#include "ManualAkteur.cuh"
#include "Utils.cuh"

#include <random>
#include <algorithm>
#include <iostream>
#include <cstring>

ManualAkteur::~ManualAkteur() {
    delete schablone;
}

ManualAkteur::ManualAkteur(std::string folder, std::string fileName) {
    schablone = Template::createDefaultTemplate(folder, fileName);
}

std::pair<char, float> ManualAkteur::act(InformationSet* informationSet) {
    RoundPlayerInfo roundInfo = schablone->roundInfos.at(informationSet->round).at(informationSet->player);
    BucketFunction* bucketFunction = roundInfo.bucketFunction;
    vector<char> bucket = bucketFunction->getBucket(informationSet->playerCardsVisible);
    int bucketPosition = bucketFunction->getBucketPosition(bucket);

    int max = (bucketFunction->bucketList.size()) / (bucketFunction->size * 2);
    bool newBucket = bucketPosition >= max;

    if (newBucket) {
        return std::pair<char, float>('f', 0.f);
    }
    else {

        std::pair<int, GameState*> result = getCurrentNode(schablone, informationSet->actionHistory);
        int currentNodeWorklistIndex = result.first;

        if (currentNodeWorklistIndex < schablone->structureList->numStateNodes) {

            std::vector<std::pair<char, float>> currentActions = result.second->getActions();

            std::cout << "Karten: ";
            for (int i = 0; i < informationSet->playerCardsVisible.size(); i++) {
                std::cout << informationSet->playerCardsVisible.at(i);
            }
            std::cout << std::endl;

            std::cout << "Gegner Action: " << informationSet->actionHistory.back().first << std::endl;

            std::cout << "Possible Actions: ";
            for (int i = 0; i < currentActions.size(); i++) {
                std::cout << currentActions.at(i).first << "(" << i << ") ";
            }
            std::cout << std::endl;
            int action;
            std::cin >> action;

            delete result.second;
            return currentActions.at(action);

        }
        else {
            // TODO
            // hier ist eigentlich schon spielende
        }

    }

    return std::pair<char, float>('O', 0.f);
}