#include "hip/hip_runtime.h"
#include "BlueprintAkteur.cuh"
#include "Utils.cuh"

#include <random>
#include <algorithm>
#include <iostream>
#include <cstring>

BlueprintAkteur::~BlueprintAkteur() {
    delete schablone;
}

BlueprintAkteur::BlueprintAkteur(std::string path) {
    schablone = Template::createDefaultTemplate(path);
}

std::pair<char, float> BlueprintAkteur::act(InformationSet* informationSet) {
    RoundPlayerInfo roundInfo = schablone->roundInfos.at(informationSet->round).at(informationSet->player);
    BucketFunction* bucketFunction = roundInfo.bucketFunction;
    vector<char> bucket = bucketFunction->getBucket(informationSet->playerCardsVisible);
    int bucketPosition = bucketFunction->getBucketPosition(bucket);

    int max = (bucketFunction->bucketList.size()) / (bucketFunction->size * 2);
    bool newBucket = bucketPosition >= max;
    int size = roundInfo.elementSize;

    if (newBucket) {
        return std::pair<char, float>('f', 0.f);
    }
    else {
        float* reads = roundInfo.blueprintHandler->readPolicies(bucketPosition, size * sizeof(float));
        std::memcpy(schablone->cumulativeRegrets.at(informationSet->player) + roundInfo.startPointTemplate, reads, size * sizeof(float));
        delete[] reads;

        std::pair<int, GameState*> result = getCurrentNode(schablone, informationSet->actionHistory);
        int currentNodeWorklistIndex = result.first;
        GameState* currentGameState = result.second;

        if (currentNodeWorklistIndex < schablone->structureList->numStateNodes) {
            TrainingInitStruct* trainingInitStruct = initTrainingInitStruct(schablone, currentNodeWorklistIndex);

            vector<float> actions(trainingInitStruct->policy, trainingInitStruct->policy + trainingInitStruct->numChildren);

            free(trainingInitStruct->policy);

            float randomNumber = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            float barrier = 0.f;
            int actionInt = trainingInitStruct->numChildren - 1;
            for (int i = 0; i < trainingInitStruct->numChildren; i++) {
                //setze barrier
                barrier += actions.at(i);
                if (barrier >= randomNumber) {
                    actionInt = i;
                    break;
                }
            }

            std::vector<std::pair<char, float>> currentActions = currentGameState->getActions();
            delete currentGameState;

            free(trainingInitStruct);
            return currentActions.at(actionInt);
        }
        else {
            // TODO
            // hier ist eigentlich schon spielende
        }

    }

    return std::pair<char, float>('O', 0.f);
}