#include "hip/hip_runtime.h"
#include "BlueprintHandler.cuh"

#include <iostream>

std::string getPath(int round, int player) {
    return "blueprint" + std::to_string(round) + std::to_string(player);
}

BlueprintHandler::BlueprintHandler(int round, int player) {
    path = getPath(round, player);
    if (!blueprintExists(path)) {
        createBlueprint(path);
    }

    std::ofstream pOfStream(path, std::ios_base::binary | std::ios_base::out | std::ios_base::in);
    std::ifstream pIfStream(path, std::ios::binary | std::ios::in);
    ofStream = std::move(pOfStream);
    ifStream = std::move(pIfStream);
}

bool BlueprintHandler::blueprintExists(std::string path) {
    std::ifstream f(path.c_str());
    return f.good();
}

void BlueprintHandler::createBlueprint(std::string path) {
    char* empty = {};

    std::ofstream out(path);
    out.write(empty, 0);
    out.close();
}


std::vector<float> BlueprintHandler::readPolicies(int pos, int size) {
    if (!ifStream.is_open()) {
        std::cout << "Input stream nicht mehr offen!" << std::endl;
    }

    char* buffer = new char[size];
    ifStream.seekg(pos * size);
    ifStream.read(buffer, size);

    float* policies = (float*)buffer;

    return std::vector<float>(policies, policies + size);
}

/**
 * REQUIRE: Wenn f�r einen Spieler eine neue policy geschrieben wird, muss diese auch f�r den anderen Spieler geschrieben werden (geg. dann leer), da die Bucketfunktion diesselbe ist -> selber Index.
 */
void BlueprintHandler::writePolicies(int pos, int size, float* policies) {
    if (!ofStream.is_open()) {
        std::cout << "Output stream nicht mehr offen!" << std::endl;
    }

    ofStream.seekp(pos * size, std::ios_base::beg);

    char* charPolicies = (char*)policies;

    ofStream.write(charPolicies, size);

    ofStream.flush();
}