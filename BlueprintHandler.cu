#include "hip/hip_runtime.h"
#include "BlueprintHandler.cuh"


#include "Logger.cuh"
#include <iostream>

std::string getPath(std::string folder, std::string fileName, int round, int player) {
    return folder + "/" + fileName + std::to_string(round) + std::to_string(player);
}

BlueprintHandler::BlueprintHandler(std::string folder, std::string fileName, int round, int player) {
    path = getPath(folder, fileName, round, player);
    if (!blueprintExists(path)) {
        createBlueprint(path);
    }

    std::ofstream pOfStream(path, std::ios_base::binary | std::ios_base::out | std::ios_base::in);
    std::ifstream pIfStream(path, std::ios::binary | std::ios::in);
    ofStream = std::move(pOfStream);
    ifStream = std::move(pIfStream);
}

bool BlueprintHandler::blueprintExists(string path) {
    std::ifstream f(path);
    return f.good();
}

void BlueprintHandler::createBlueprint(string path) {
    char* empty = {};

    std::ofstream out(path);
    out.write(empty, 0);
    out.close();
}


float* BlueprintHandler::readPolicies(int pos, int size) {
    if (!ifStream.is_open()) {
        Logger::throwRuntimeError("Input stream nicht mehr offen!");
    }

    char* buffer = new char[size];
    ifStream.seekg(pos * size);
    ifStream.read(buffer, size);

    float* policies = (float*)buffer;

    return policies;
}

/**
 * REQUIRE: Wenn f�r einen Spieler eine neue policy geschrieben wird, muss diese auch f�r den anderen Spieler geschrieben werden (geg. dann leer), da die Bucketfunktion diesselbe ist -> selber Index.
 */
void BlueprintHandler::writePolicies(int pos, int size, float* policies) {
    if (!ofStream.is_open()) {
        Logger::throwRuntimeError("Output stream nicht mehr offen!");
    }

    ofStream.seekp(pos * size, std::ios_base::beg);

    char* charPolicies = (char*)policies;

    ofStream.write(charPolicies, size);

    ofStream.flush();
}

std::string BlueprintHandler::getFileSize() {
    std::ifstream file(path, std::ios::binary | std::ios::ate);
    return std::to_string(file.tellg());
}