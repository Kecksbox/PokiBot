#include "hip/hip_runtime.h"
#include "BucketFunction.cuh"
#include <algorithm>
#include <iterator>
#include <map>
#include <set>
#include <list>
#include <iostream>
#include <fstream>
#include <cstdio>

using std::string;
using std::vector;

std::map<char, int> ranksMap = {
        {'2',0},{'3',1},{'4',2},{'5',3},{'6',4},{'7',5},{'8',6},{'9',7},{'T',8},{'J',9},{'Q',10},{'K',11},{'A',12}
};

BucketFunction::BucketFunction(std::string folder, std::string fileName, int round, size_t size) {
    this->folder = folder,
    this->fileName = fileName;
    this->round = round;
    //Anzahl der Karten, nicht Anzahl der Chars
    this->size = size;

    loadBucketFunction();
}

void BucketFunction::loadBucketFunction() {
    std::ifstream t;
    int length;
    t.open(folder + "/" + fileName);      // open input file

    if (!t.good()) {
        //No file found
    }
    else {
        t.seekg(0, std::ios::end);    // go to the end
        length = t.tellg();           // report location (this is the length)
        t.seekg(0, std::ios::beg);    // go back to the beginning
        char* buffer = new char[length];    // allocate memory for a buffer of appropriate dimension
        t.read(buffer, length);       // read the whole file into the buffer
        t.close();

        bucketList.assign(buffer, buffer + length);
        delete[] buffer;
    }
}

void BucketFunction::saveBucketFunction() {
    if (bucketList.empty()) {
        throw "bucketList is empty despite the training being over";
    }
    char* bucketListArray = &bucketList[0];

    std::ofstream out(folder + "/" + fileName);
    out.write(bucketListArray, bucketList.size() * sizeof(char));
    out.close();
}

std::map<char, char> rankToClassified = {
       {'2','A'},{'3','A'},{'4','A'},{'5','B'},{'6','B'},{'7','B'},{'8','C'},{'9','C'},{'T','C'},{'J','D'},{'Q','D'},{'K','E'},{'A','E'}
};

char indexToRank[13] = { '2', '3', '4', '5', '6', '7', '8', '9', 'T', 'J', 'Q', 'K', 'A' };

std::vector<char> BucketFunction::bucketCardsToNumbersNeglectStreetsAndFlushes(std::vector<std::string>* cards) {
    //high card = drei klassen (A, B, C, D,  E)
    //Paare genauso
    //H: {A, K, Q, J}
    //M: {T, 9, 8}
    //L:{2, 3, 4, 5, 6, 7}

    std::map<char, int> rankCount = {
       {'2',0},{'3',0},{'4',0},{'5',0},{'6',0},{'7',0},{'8',0},{'9',0},{'T',0},{'J',0},{'Q',0},{'K',0},{'A',0}
    };

    std::map<char, int> suitCount = {
        {'c',0}, {'s',0}, {'h',0}, {'d',0}
    };

    for (int i = 0; i < size; i++) {
        rankCount.at(cards->at(i).at(0))++;
        suitCount.at(cards->at(i).at(1))++;
    }
    int maxSuit = std::max({ suitCount.at('c'),suitCount.at('s'),suitCount.at('h'),suitCount.at('d') });

    std::vector<char> bucket;
    bucket.reserve(size * (size_t)2);

    std::string bucketString;

    for (int i = 12; i >= 0; i--) {
        char rank = indexToRank[i];
        int count = rankCount.at(rank);
        if (count == 4) {
            char classifier = rankToClassified.at(rank);
            std::string s(1, classifier);
            //W = Platzwalter
            bucketString = "P4" + s + "W" + "WWWW";
            bucket.insert(bucket.end(), bucketString.begin(), bucketString.end());
        }
    }

    for (int i = 12; i >= 0; i--) {
        char rank = indexToRank[i];
        int count = rankCount.at(rank);
        if (count == 3) {
            char classifier = rankToClassified.at(rank);
            std::string s(1, classifier);
            //W = Platzwalter
            bucketString = "P3" + s + "W" + "WW";
            bucket.insert(bucket.end(), bucketString.begin(), bucketString.end());
        }
    }

    for (int i = 12; i >= 0; i--) {
        char rank = indexToRank[i];
        int count = rankCount.at(rank);
        if (count == 2) {
            char classifier = rankToClassified.at(rank);
            std::string s(1, classifier);
            //W = Platzwalter
            bucketString = "P2" + s + "W";
            bucket.insert(bucket.end(), bucketString.begin(), bucketString.end());
        }
    }

    for (int i = 12; i >= 0; i--) {
        char rank = indexToRank[i];
        int count = rankCount.at(rank);
        if (count == 1) {
            char classifier = rankToClassified.at(rank);
            std::string s(1, classifier);
            //W = Platzwalter
            bucketString = s + "W";
            bucket.insert(bucket.end(), bucketString.begin(), bucketString.end());
        }
    }

    if (round == 1) {
        if (maxSuit >= 3) bucket.back() = maxSuit;
    }
    else if (round == 2) {
        if (maxSuit >= 4) bucket.back() = maxSuit;
    }
    else if (round == 3) {
        if (maxSuit >= 5) bucket.back() = 5;
    }

    return bucket;
}

std::vector<char> BucketFunction::getBucket(std::vector<std::string> cards) {
    return bucketCardsToNumbersNeglectStreetsAndFlushes(&cards);
}

int BucketFunction::getBucketPosition(std::vector<char> bucket) {
    int i = 0;
    for (; i < bucketList.size() / (size * 2); i++) {
        bool matched = true;
        for (int j = 0; j < size * 2; j++) {
            if (bucket[j] != bucketList.at((i * size * 2) + j)) {
                matched = false;
                break;
            }
        }
        if (matched) return i;
    }
    return i;
}