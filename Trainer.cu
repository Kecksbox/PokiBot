#include "hip/hip_runtime.h"
#include "Trainer.cuh"
#include "Template.cuh"
#include "Logger.cuh"
#include "Cards.cuh"
#include "Utils.cuh"

#include "SolverA.cuh"

#include <random>
#include <algorithm>
#include <numeric>
#include <cstring>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include "hip/device_functions.h"
#include ""

#include <stdio.h>

TexasHoldemTrainer::~TexasHoldemTrainer() {
    delete schablone;
}

TexasHoldemTrainer::TexasHoldemTrainer(std::string path) {
    blueprintHandler = nullptr;
    schablone = Template::createDefaultTemplate(path);
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


DeviceStructureList* prepareDevice(Template* schablone) {
    int numStateNodes = schablone->structureList->numStateNodes;
    int numLeafNodes = schablone->structureList->numLeafNodes;
    int worklistSize = numStateNodes + numLeafNodes;

    DeviceStructureList* dsl = new DeviceStructureList();
    size_t size = 0;

    size = sizeof(int) * numStateNodes;
    gpuErrchk(hipMalloc((void**)&dsl->childrenWorklistPointers, size));
    gpuErrchk(hipMemcpy(dsl->childrenWorklistPointers, schablone->structureList->childrenWorklistPointers, size, hipMemcpyHostToDevice));

    size = sizeof(bool) * numLeafNodes;
    gpuErrchk(hipMalloc((void**)&dsl->folded, size));
    gpuErrchk(hipMemcpy(dsl->folded, schablone->structureList->folded, size, hipMemcpyHostToDevice));

    size = sizeof(int) * numStateNodes;
    gpuErrchk(hipMalloc((void**)&dsl->numChildren, size));
    gpuErrchk(hipMemcpy(dsl->numChildren, schablone->structureList->numChildren, size, hipMemcpyHostToDevice));

    size = sizeof(int);
    gpuErrchk(hipMalloc((void**)&dsl->numLeafNodes, size));
    gpuErrchk(hipMemcpy(dsl->numLeafNodes, &schablone->structureList->numLeafNodes, size, hipMemcpyHostToDevice));

    size = sizeof(int);
    gpuErrchk(hipMalloc((void**)&dsl->numStateNodes, size));
    gpuErrchk(hipMemcpy(dsl->numStateNodes, &schablone->structureList->numStateNodes, size, hipMemcpyHostToDevice));

    size = sizeof(float) * worklistSize;
    gpuErrchk(hipMalloc((void**)&dsl->payoff, size));
    
    size = sizeof(bool) * worklistSize;
    gpuErrchk(hipMalloc((void**)&dsl->player0, size));
    gpuErrchk(hipMemcpy(dsl->player0, schablone->structureList->player0, size, hipMemcpyHostToDevice));

    size = sizeof(int) * numStateNodes;
    gpuErrchk(hipMalloc((void**)&dsl->policyPointers, size));
    gpuErrchk(hipMemcpy(dsl->policyPointers, schablone->structureList->policyPointers, size, hipMemcpyHostToDevice));

    size = sizeof(float) * numLeafNodes * 2;
    gpuErrchk(hipMalloc((void**)&dsl->pots, size));
    gpuErrchk(hipMemcpy(dsl->pots, schablone->structureList->pots, size, hipMemcpyHostToDevice));

    size = sizeof(float) * numStateNodes * 2;
    gpuErrchk(hipMalloc((void**)&dsl->reachProbabilities, size));

    size = sizeof(int) * worklistSize;
    gpuErrchk(hipMalloc((void**)&dsl->worklist, size));
    gpuErrchk(hipMemcpy(dsl->worklist, schablone->structureList->worklist, size, hipMemcpyHostToDevice));

    size = sizeof(bool);
    gpuErrchk(hipMalloc((void**)&dsl->playerWon, size));

    size = sizeof(bool);
    gpuErrchk(hipMalloc((void**)&dsl->draw, size));

    size = sizeof(int);
    gpuErrchk(hipMalloc((void**)&dsl->levelStart, size));

    size = sizeof(int);
    gpuErrchk(hipMalloc((void**)&dsl->numElements, size));

    size = schablone->roundInfos.at(3).at(0).startPointTemplate + schablone->roundInfos.at(3).at(0).elementSize;
    size = size * sizeof(float);
    gpuErrchk(hipMalloc((void**)&dsl->cumulativeRegrets0, size));
    gpuErrchk(hipMalloc((void**)&dsl->policy0, size));

    size = schablone->roundInfos.at(3).at(1).startPointTemplate + schablone->roundInfos.at(3).at(1).elementSize;
    size = size * sizeof(float);
    gpuErrchk(hipMalloc((void**)&dsl->cumulativeRegrets1, size));
    gpuErrchk(hipMalloc((void**)&dsl->policy1, size));

    size = sizeof(DeviceStructureList);
    gpuErrchk(hipMalloc((void**)&dsl->Dself, size));
    gpuErrchk(hipMemcpy(dsl->Dself, dsl, size, hipMemcpyHostToDevice));

    return dsl;
}


void freeDeviceStructureList(DeviceStructureList* dsl) {
    gpuErrchk(hipFree(dsl->childrenWorklistPointers));
    gpuErrchk(hipFree(dsl->folded));
    gpuErrchk(hipFree(dsl->numChildren));
    gpuErrchk(hipFree(dsl->numLeafNodes));
    gpuErrchk(hipFree(dsl->numStateNodes));
    gpuErrchk(hipFree(dsl->payoff));
    gpuErrchk(hipFree(dsl->player0));
    gpuErrchk(hipFree(dsl->policyPointers));
    gpuErrchk(hipFree(dsl->pots));
    gpuErrchk(hipFree(dsl->reachProbabilities));
    gpuErrchk(hipFree(dsl->worklist));

    gpuErrchk(hipFree(dsl->playerWon));
    gpuErrchk(hipFree(dsl->draw));

    gpuErrchk(hipFree(dsl->levelStart));
    gpuErrchk(hipFree(dsl->numElements));

    gpuErrchk(hipFree(dsl->cumulativeRegrets0));
    gpuErrchk(hipFree(dsl->cumulativeRegrets1));
    gpuErrchk(hipFree(dsl->policy0));
    gpuErrchk(hipFree(dsl->policy1));

    gpuErrchk(hipFree(dsl->Dself));
    free(dsl);
}


void TexasHoldemTrainer::trainSequentiell(int numIterations, bool useGpu) {
    Logger::logToConsole("Training start");
    vector<string> cards;
    cards.reserve(52);
    vector<string> player0Cards;
    player0Cards.reserve(7);
    vector<string> player1Cards;
    player1Cards.reserve(7);
    vector<vector<string>> playerCards = { player0Cards, player1Cards };

    DeviceStructureList* deviceStructureListPtr = nullptr;
    if (useGpu) {
        deviceStructureListPtr = prepareDevice(this->schablone);
    }

    for (int i = 0; i < numIterations; i++) {
        cards = getCards();
        playerCards.at(0) = { cards.at(0), cards.at(1), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8) };
        playerCards.at(1) = { cards.at(2), cards.at(3), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8) };

        if (i % 1000 == 0) {
            Logger::logIteration(i);
        }
        
        if (useGpu) {
            trainGPU(&playerCards, deviceStructureListPtr);
        }
        else {
            trainCPU(&playerCards);
        }
    }

    if (useGpu) {
        freeDeviceStructureList(deviceStructureListPtr);
    }

    for (int round = 0; round < 4; round++) {
        schablone->roundInfos.at(round).at(0).bucketFunction->saveBucketFunction();
    }
}

void TexasHoldemTrainer::trainCPU(vector<vector<string>>* playerCards) {

    //a) bestimme gewinner
    int player0Eval = test7(playerCards->at(0));
    int player1Eval = test7(playerCards->at(1));

    bool draw = player0Eval == player1Eval;
    bool playerWon = player0Eval > player1Eval;

    //b) setze payoffs in leafs durch gewinner
    int numLeafNodes = schablone->structureList->numLeafNodes;
    int numStateNodes = schablone->structureList->numStateNodes;
    
    for (int i = 0; i < numLeafNodes; i++) {
        float* potPointer = schablone->structureList->pots + i * (size_t)2;
        bool folded = schablone->structureList->folded[i];
        bool player0 = schablone->structureList->player0[numStateNodes + i];
        bool currentPlayer = player0 ? 0 : 1;

        bool localePlayerWon = playerWon;
        if (folded) {
            localePlayerWon = currentPlayer;
        }

        float payoff = 0.f;
        if (!draw) {
            payoff = potPointer[(currentPlayer + 1) % 2];
        }

        schablone->structureList->payoff[numStateNodes + i] = (localePlayerWon == currentPlayer ? payoff : -payoff);
    }

    //c_1) prepare strategie laden
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            vector<char> bucket = info.bucketFunction->getBucket(playerCards->at(player));
            int pos = info.bucketFunction->getBucketPosition(bucket);
            int size = info.elementSize;

            int max = (info.bucketFunction->bucketList.size()) / (info.bucketFunction->size * 2);
            bool newBucket = pos >= max;

            if (newBucket) {
                info.bucketFunction->bucketList.insert(info.bucketFunction->bucketList.end(), bucket.begin(), bucket.end());
                float* zeroArray = new float[size] {0.f};
                info.blueprintHandler->writePolicies(pos, size * sizeof(float), zeroArray);
                delete[](zeroArray);

                int otherPlayer = (player + 1) % 2;
                RoundPlayerInfo otherInfo = schablone->roundInfos.at(round).at(otherPlayer);
                int otherSize = otherInfo.elementSize;
                float* otherZeroArray = new float[otherSize] {0.f};
                otherInfo.blueprintHandler->writePolicies(pos, otherSize * sizeof(float), otherZeroArray);
                delete[](otherZeroArray);
            }
            float* reads = info.blueprintHandler->readPolicies(pos, size * sizeof(float));
            std::memcpy(schablone->cumulativeRegrets.at(player) + info.startPointTemplate, reads, size * sizeof(float));
            delete[] reads;
        }
    }

    //c_2) forwardpass: setze reach probabilities
    schablone->structureList->reachProbabilities[0] = 1.f;
    schablone->structureList->reachProbabilities[1] = 1.f;

    for (int i = 0; i < numStateNodes; i++) {
        TrainingInitStruct* trainingInitStruct = initTrainingInitStruct(schablone, i);

        int numChildren = trainingInitStruct->numChildren;
        int* children = trainingInitStruct->children;
        int otherPlayer = trainingInitStruct->otherPlayer;
        int currentPlayer = trainingInitStruct->currentPlayer;
        float* policy = trainingInitStruct->policy;
        float* reachProbabilitiesLocal = trainingInitStruct->reachProbabilitiesLocal;

        for (int j = 0; j < numChildren; j++) {
            if (children[j] < numStateNodes) {
                schablone->structureList->reachProbabilities[2 * children[j] + currentPlayer] = policy[j] * reachProbabilitiesLocal[currentPlayer];
                schablone->structureList->reachProbabilities[2 * children[j] + otherPlayer] = reachProbabilitiesLocal[otherPlayer];
            }
        }
        free(trainingInitStruct->policy);
        free(trainingInitStruct);
    }

    //d_1) backwardpass: setze regrets
    for (int i = numStateNodes - 1; i >= 0; i--) {
        TrainingInitStruct* trainingInitStruct = initTrainingInitStruct(schablone, i);

        int* children = trainingInitStruct->children;
        int numChildren = trainingInitStruct->numChildren;

        vector<float> upstreamPayoffs;
        upstreamPayoffs.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            upstreamPayoffs.push_back(-1 * schablone->structureList->payoff[children[j]]);
        }

        float* policy = trainingInitStruct->policy;

        float* cumulativeRegrets = trainingInitStruct->cumulativeRegrets;

        float nodeUtility = std::inner_product(policy, policy + numChildren, upstreamPayoffs.begin(), 0.f);
        schablone->structureList->payoff[i] = nodeUtility;

        float* reachProbabilitiesLocal = trainingInitStruct->reachProbabilitiesLocal;
        int currentPlayer = trainingInitStruct->currentPlayer;
        int otherPlayer = trainingInitStruct->otherPlayer;

        float counterValue = reachProbabilitiesLocal[currentPlayer] * nodeUtility;

        vector<float> counterActionValues;
        counterActionValues.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            counterActionValues.push_back(1 * upstreamPayoffs[j]);
        }

        vector<float> counterRegrets;
        counterRegrets.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            counterRegrets.push_back(reachProbabilitiesLocal[otherPlayer] * (counterActionValues[j] - counterValue));

            cumulativeRegrets[j] = cumulativeRegrets[j] + std::max(0.f, counterRegrets[j]);
        }
        free(trainingInitStruct->policy);
        free(trainingInitStruct);
    }

    //d_2) postpare strategie zur�ckschreiben
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            int pos = info.bucketFunction->getBucketPosition(info.bucketFunction->getBucket(playerCards->at(player)));

            int size = info.elementSize;

            info.blueprintHandler->writePolicies(pos, size * sizeof(float), schablone->cumulativeRegrets.at(player) + info.startPointTemplate);
        }
    }
}

__global__ void setLeafPayoffs(DeviceStructureList* dsl) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= *dsl->numElements) {
        return;
    }

    float* potPointer = dsl->pots + id * 2;
    bool localFolded = dsl->folded[id];
    bool localPlayer0 = dsl->player0[*dsl->numStateNodes + id];
    bool currentPlayer = localPlayer0 ? 0 : 1;

    bool localPlayerWon = *dsl->playerWon;
    if (localFolded) {
        localPlayerWon = currentPlayer;
    }

    float localPayoff = 0.f;
    if (!*dsl->draw) {
        localPayoff = potPointer[1 - currentPlayer];
    }

    dsl->payoff[*dsl->numStateNodes + id] = (localPlayerWon == currentPlayer ? localPayoff : -localPayoff);
}

__device__ TrainingInitStruct* getInitStructGPU(DeviceStructureList* dsl, int i) {
    int policyPointer = dsl->policyPointers[i];
    int numChildren = dsl->numChildren[i];
    int childrenWorklistPointer = dsl->childrenWorklistPointers[i];

    int currentPlayer = dsl->player0[i] ? 0 : 1;
    float* cummulativeRegrets = dsl->player0[i] ? dsl->cumulativeRegrets0 + policyPointer : dsl->cumulativeRegrets1 + policyPointer;

    float* policy = dsl->player0[i] ? dsl->policy0 + policyPointer : dsl->policy1 + policyPointer;

    float* reachProbabilitiesLocal = dsl->reachProbabilities + (i * 2);
    int* children = dsl->worklist + childrenWorklistPointer;
    int otherPlayer = 1 - currentPlayer;

    auto trainingInitStruct = new TrainingInitStruct();

    trainingInitStruct->policyPointer = policyPointer;
    trainingInitStruct->numChildren = numChildren;
    trainingInitStruct->childrenWorklistPointer = childrenWorklistPointer;
    trainingInitStruct->currentPlayer = currentPlayer;
    trainingInitStruct->cumulativeRegrets = cummulativeRegrets;
    trainingInitStruct->policy = policy;
    trainingInitStruct->reachProbabilitiesLocal = reachProbabilitiesLocal;
    trainingInitStruct->children = children;
    trainingInitStruct->otherPlayer = otherPlayer;

    return trainingInitStruct;
}

__global__ void setReachProbsAndPolicy(DeviceStructureList* dsl) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= *dsl->numElements) {
        return;
    }

    id += *dsl->levelStart;

    auto tis = getInitStructGPU(dsl, id);

    int numChildren = tis->numChildren;
    float* policy = tis->policy;
    int* children = tis->children;
    int currentPlayer = tis->currentPlayer;
    int otherPlayer = tis->otherPlayer;
    float* reachProbabilitiesLocal = tis->reachProbabilitiesLocal;

    for (int i = 0; i < numChildren; i++) {
        policy[i] = fmaxf(policy[i], 0.f);
    }

    float arraySum = 0;
    for (int i = 0; i < numChildren; i++) {
        arraySum += policy[i];
    }

    if (arraySum > 0) {
        for (int i = 0; i < numChildren; i++) {
            policy[i] /= arraySum;
        }
    }
    else {
        for (int i = 0; i < numChildren; i++) {
            policy[i] = 1.f / numChildren;
        }
    }

    for (int j = 0; j < numChildren; j++) {
        if (children[j] < *dsl->numStateNodes) {
            dsl->reachProbabilities[2 * children[j] + currentPlayer] = policy[j] * reachProbabilitiesLocal[currentPlayer];
            dsl->reachProbabilities[2 * children[j] + otherPlayer] = reachProbabilitiesLocal[otherPlayer];
        }
    }

    free(tis);
}

__global__ void setRegrets(DeviceStructureList* dsl) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= *dsl->numElements) {
        return;
    }

    id += *dsl->levelStart;

    TrainingInitStruct* trainingInitStruct = getInitStructGPU(dsl, id);

    int* children = trainingInitStruct->children;
    int numChildren = trainingInitStruct->numChildren;

    float* upstreamPayoffs = new float[numChildren];

    float nodeUtility = 0.f;
    for (int j = 0; j < numChildren; j++) {

        upstreamPayoffs[j] = -dsl->payoff[children[j]];

        nodeUtility += trainingInitStruct->policy[j] * upstreamPayoffs[j];

    }

    dsl->payoff[id] = nodeUtility;

    float* reachProbabilitiesLocal = trainingInitStruct->reachProbabilitiesLocal;
    int currentPlayer = trainingInitStruct->currentPlayer;
    int otherPlayer = trainingInitStruct->otherPlayer;

    float counterValue = reachProbabilitiesLocal[currentPlayer] * nodeUtility;

    float* cumulativeRegrets = trainingInitStruct->cumulativeRegrets;
    for (int j = 0; j < numChildren; j++) {
        float counterActionValue = upstreamPayoffs[j];
        cumulativeRegrets[j] = cumulativeRegrets[j] + fmaxf(0.f, reachProbabilitiesLocal[otherPlayer] * (counterActionValue - counterValue));
    }

    free(upstreamPayoffs);

    free(trainingInitStruct);
}

struct GetIndexReturnType {
    int levelStart = 0;
    int numElements = 0;
};

GetIndexReturnType getIndexList(Template* schablone, int levelIndex) {
    auto levelPointers = schablone->structureList->levelPointers;
    int levelStart = schablone->structureList->worklist[levelPointers.at(levelIndex)];
    int numElements;
    if (levelIndex == levelPointers.size() - 1) {
        numElements = schablone->structureList->numStateNodes - levelStart;
    }
    else {
        numElements = schablone->structureList->worklist[levelPointers.at(levelIndex + 1)] - levelStart;
    }

    return GetIndexReturnType { levelStart, numElements};
}

void TexasHoldemTrainer::trainGPU(vector<vector<string>>* playerCards, DeviceStructureList* dsl) {

    //a) bestimme gewinner
    int player0Eval = test7(playerCards->at(0));
    int player1Eval = test7(playerCards->at(1));

    bool draw = player0Eval == player1Eval;
    bool playerWon = player0Eval > player1Eval;
    gpuErrchk(hipMemcpy(dsl->draw, &draw, sizeof(bool), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dsl->playerWon, &playerWon, sizeof(bool), hipMemcpyHostToDevice));

    //b) setze payoffs in leafs durch gewinner
    int numLeafNodes = schablone->structureList->numLeafNodes;

    int N = numLeafNodes;
    hipMemcpy(dsl->numElements, &N, sizeof(int), hipMemcpyHostToDevice);
    int blockSize = BLOCKSIZE;
    int numBlocks = (N + blockSize - 1) / blockSize;
    setLeafPayoffs << < numBlocks, blockSize >> > (dsl->Dself);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    //c_1) prepare strategie laden
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            vector<char> bucket = info.bucketFunction->getBucket(playerCards->at(player));
            int pos = info.bucketFunction->getBucketPosition(bucket);
            int size = info.elementSize;

            int max = (info.bucketFunction->bucketList.size()) / (info.bucketFunction->size * 2);
            bool newBucket = pos >= max;

            if (newBucket) {
                info.bucketFunction->bucketList.insert(info.bucketFunction->bucketList.end(), bucket.begin(), bucket.end());
                float* zeroArray = new float[size] {0.f};
                info.blueprintHandler->writePolicies(pos, size * sizeof(float), zeroArray);
                delete[](zeroArray);

                int otherPlayer = (player + 1) % 2;
                RoundPlayerInfo otherInfo = schablone->roundInfos.at(round).at(otherPlayer);
                int otherSize = otherInfo.elementSize;
                float* otherZeroArray = new float[otherSize] {0.f};
                otherInfo.blueprintHandler->writePolicies(pos, otherSize * sizeof(float), otherZeroArray);
                delete[](otherZeroArray);
            }
            float* reads = info.blueprintHandler->readPolicies(pos, size * sizeof(float));

            size = size * sizeof(float);
            if (player == 0) {
                gpuErrchk(hipMemcpy(dsl->cumulativeRegrets0 + info.startPointTemplate, reads, size, hipMemcpyHostToDevice));
                gpuErrchk(hipMemcpy(dsl->policy0 + info.startPointTemplate, reads, size, hipMemcpyHostToDevice));
            }
            else {
                gpuErrchk(hipMemcpy(dsl->cumulativeRegrets1 + info.startPointTemplate, reads, size, hipMemcpyHostToDevice));
                gpuErrchk(hipMemcpy(dsl->policy1 + info.startPointTemplate, reads, size, hipMemcpyHostToDevice));
            }

            delete[] reads;
        }
    }

    hipDeviceSynchronize();

    //c_2) forwardpass: setze reach probabilities
    schablone->structureList->reachProbabilities[0] = 1.f;
    schablone->structureList->reachProbabilities[1] = 1.f;

    gpuErrchk(hipMemcpy(dsl->reachProbabilities, schablone->structureList->reachProbabilities, sizeof(float) * 2, hipMemcpyHostToDevice));

    auto levelPointers = schablone->structureList->levelPointers;
    for (int i = 0; i < levelPointers.size(); i++) {
        GetIndexReturnType indexListData = getIndexList(schablone, i);
        
        int numElements = indexListData.numElements;
        hipMemcpy(dsl->levelStart, &indexListData.levelStart, sizeof(int), hipMemcpyHostToDevice);

        int N = numElements;
        hipMemcpy(dsl->numElements, &N, sizeof(int), hipMemcpyHostToDevice);
        int blockSize = BLOCKSIZE;
        int numBlocks = (N + blockSize - 1) / blockSize;
        setReachProbsAndPolicy << < numBlocks, blockSize >> > (dsl->Dself);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        hipDeviceSynchronize();
    }

    //d_1) backwardpass: setze regrets
    for (int i = levelPointers.size() - 1; i >= 0; i--) {
        GetIndexReturnType indexListData = getIndexList(schablone, i);

        int numElements = indexListData.numElements;
        hipMemcpy(dsl->levelStart, &indexListData.levelStart, sizeof(int), hipMemcpyHostToDevice);

        int N = numElements;
        hipMemcpy(dsl->numElements, &N, sizeof(int), hipMemcpyHostToDevice);
        int blockSize = BLOCKSIZE;
        int numBlocks = (N + blockSize - 1) / blockSize;
        setRegrets << < numBlocks, blockSize >> > (dsl->Dself);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        hipDeviceSynchronize();
    }

    //d_2) postpare strategie zur�ckschreiben

    int dArrSize = schablone->roundInfos.at(3).at(0).startPointTemplate + schablone->roundInfos.at(3).at(0).elementSize;
    dArrSize = dArrSize * sizeof(float);
    hipMemcpy(schablone->cumulativeRegrets.at(0), dsl->cumulativeRegrets0, dArrSize, hipMemcpyDeviceToHost);
    dArrSize = schablone->roundInfos.at(3).at(1).startPointTemplate + schablone->roundInfos.at(3).at(1).elementSize;
    dArrSize = dArrSize * sizeof(float);
    hipMemcpy(schablone->cumulativeRegrets.at(1), dsl->cumulativeRegrets1, dArrSize, hipMemcpyDeviceToHost);

    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            int pos = info.bucketFunction->getBucketPosition(info.bucketFunction->getBucket(playerCards->at(player)));

            int size = info.elementSize;

            info.blueprintHandler->writePolicies(pos, size * sizeof(float), schablone->cumulativeRegrets.at(player) + info.startPointTemplate);
        }
    }
}