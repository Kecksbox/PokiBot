#include "hip/hip_runtime.h"
#include "Trainer.cuh"
#include "Template.cuh"
#include "Logger.cuh"
#include "Cards.cuh"
#include "Utils.cuh"

#include <random>
#include <algorithm>
#include <numeric>
#include <cstring>

using std::vector;

TexasHoldemTrainer::~TexasHoldemTrainer() {
    delete schablone;
}

TexasHoldemTrainer::TexasHoldemTrainer(std::string path) {
    blueprintHandler = nullptr;
    schablone = Template::createDefaultTemplate(path);
}

int TexasHoldemTrainer::trainSequentiell(int numIterations) {
    Logger::log("Training start");

    int util = 0;
    vector<string> cards;
    cards.reserve(52);
    vector<string> player0Cards;
    player0Cards.reserve(7);
    vector<string> player1Cards;
    player1Cards.reserve(7);
    vector<vector<string>> playerCards = { player0Cards, player1Cards };

    for (int i = 0; i < numIterations; i++) {
        cards = getCards();
        playerCards.at(0) = { cards.at(0), cards.at(1), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8) };
        playerCards.at(1) = { cards.at(2), cards.at(3), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8) };

        if (i % 1000 == 0) {
            std::cout << "train " << i << std::endl;
        }

        util += train(&playerCards);
    }

    std::cout << "saveBucketFunctions" << std::endl;
    for (int round = 0; round < 4; round++) {
        schablone->roundInfos.at(round).at(0).bucketFunction->saveBucketFunction();
    }

    Logger::log("Training success");
    return util;
}

int TexasHoldemTrainer::train(vector<vector<string>>* playerCards) {

    //a) bestimme gewinner
    int player0Eval = 1;
    int player1Eval = 0;

    bool draw = player0Eval == player1Eval;
    bool playerWon = player0Eval > player1Eval;

    //b) setze payoffs in leafs durch gewinner
    int numLeafNodes = schablone->structureList->numLeafNodes;
    int numStateNodes = schablone->structureList->numStateNodes;

    for (int i = 0; i < numLeafNodes; i++) {
        vector<float> potVector(schablone->structureList->pots.begin() + i * 2, schablone->structureList->pots.begin() + i * 2 + 2);
        bool folded = schablone->structureList->folded.at(i);
        bool player0 = schablone->structureList->player0.at(numStateNodes + i);
        bool currentPlayer = player0 ? 0 : 1;

        bool localePlayerWon = playerWon;
        if (folded) {
            localePlayerWon = currentPlayer;
        }

        float payoff = 0.f;
        if (!draw) {
            payoff = potVector.at((currentPlayer + 1) % 2);
        }

        schablone->structureList->payoff.at(numStateNodes + i) = (localePlayerWon == currentPlayer ? payoff : -payoff);
    }

    //c_1) prepare strategie laden
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            vector<char> bucket = info.bucketFunction->getBucket(playerCards->at(player));
            int pos = info.bucketFunction->getBucketPosition(bucket);
            int size = info.elementSize;

            int max = (info.bucketFunction->bucketList.size()) / (info.bucketFunction->size * 2);
            bool newBucket = pos >= max;

            if (newBucket) {
                info.bucketFunction->bucketList.insert(info.bucketFunction->bucketList.end(), bucket.begin(), bucket.end());
                vector<float> zeroVector = vector<float>(size, 0.f);
                info.blueprintHandler->writePolicies(pos, size * sizeof(float), &zeroVector.at(0));

                int otherPlayer = (player + 1) % 2;
                RoundPlayerInfo otherInfo = schablone->roundInfos.at(round).at(otherPlayer);
                int otherSize = otherInfo.elementSize;
                vector<float> otherZeroVector = vector<float>(otherSize, 0.f);
                otherInfo.blueprintHandler->writePolicies(pos, otherSize * sizeof(float), &otherZeroVector.at(0));
            }
            vector<float> reads = info.blueprintHandler->readPolicies(pos, size * sizeof(float));
            std::copy_n(reads.begin(), size, schablone->cumulativeRegrets.at(player).begin() + info.startPointTemplate);
        }
    }

    //c_2) forwardpass: setze reach probabilities
    schablone->structureList->reachProbabilities.at(0) = 1.f;
    schablone->structureList->reachProbabilities.at(1) = 1.f;

    for (int i = 0; i < numStateNodes; i++) {
        std::unique_ptr<TrainingInitStruct> trainingInitStruct = initTrainingInitStruct(schablone, i);

        int numChildren = trainingInitStruct->numChildren;
        vector<int> children = trainingInitStruct->children;
        int otherPlayer = trainingInitStruct->otherPlayer;
        int currentPlayer = trainingInitStruct->currentPlayer;
        vector<float> policy = trainingInitStruct->policy;
        vector<float> reachProbabilitiesLocal = trainingInitStruct->reachProbabilitiesLocal;

        for (int j = 0; j < numChildren; j++) {
            auto test = policy.at(j);
            auto test2 = reachProbabilitiesLocal.at(currentPlayer);
            auto test3 = children.at(j);


            if (children.at(j) < numStateNodes) {
                schablone->structureList->reachProbabilities.at(2 * children.at(j) + currentPlayer) = policy.at(j) * reachProbabilitiesLocal.at(currentPlayer);
                schablone->structureList->reachProbabilities.at(2 * children.at(j) + otherPlayer) = reachProbabilitiesLocal.at(otherPlayer);
            }
              
        }
    }

    //d_1) backwardpass: setze regrets
    for (int i = numStateNodes - 1; i >= 0; i--) {
        std::unique_ptr<TrainingInitStruct> trainingInitStruct = initTrainingInitStruct(schablone, i);

        vector<int> children = trainingInitStruct->children;
        int numChildren = trainingInitStruct->numChildren;

        vector<float> upstreamPayoffs;
        upstreamPayoffs.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            upstreamPayoffs.push_back(-1 * schablone->structureList->payoff[children[j]]);
        }

        vector<float> policy = trainingInitStruct->policy;

        vector<float> cumulativeRegrets = trainingInitStruct->cumulativeRegrets;

        float nodeUtility = std::inner_product(policy.begin(), policy.begin() + numChildren, upstreamPayoffs.begin(), 0.f);
        schablone->structureList->payoff[i] = nodeUtility;

        vector<float> reachProbabilitiesLocal = trainingInitStruct->reachProbabilitiesLocal;
        int currentPlayer = trainingInitStruct->currentPlayer;
        int otherPlayer = trainingInitStruct->otherPlayer;

        float counterValue = reachProbabilitiesLocal[currentPlayer] * nodeUtility;

        vector<float> counterActionValues;
        counterActionValues.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            counterActionValues.push_back(1 * upstreamPayoffs[j]);
        }

        vector<float> counterRegrets;
        counterRegrets.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            counterRegrets.push_back(reachProbabilitiesLocal[otherPlayer] * (counterActionValues[j] - counterValue));

            cumulativeRegrets.at(j) = cumulativeRegrets.at(j) + std::max(0.f, counterRegrets[j]);
        }
    }

    //d_2) postpare strategie zur�ckschreiben
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            int pos = info.bucketFunction->getBucketPosition(info.bucketFunction->getBucket(playerCards->at(player)));

            int size = info.elementSize;

            info.blueprintHandler->writePolicies(pos, size * sizeof(float), &schablone->cumulativeRegrets.at(player).at(info.startPointTemplate));
        }
    }
    return 0;
}