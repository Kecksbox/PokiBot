#include "hip/hip_runtime.h"
#include "Trainer.cuh"
#include "Template.cuh"
#include "Logger.cuh"
#include "Cards.cuh"
#include "Utils.cuh"

#include "SolverA.cuh"

#include <random>
#include <algorithm>
#include <numeric>
#include <cstring>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include "hip/device_functions.h"
#include ""

#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

TexasHoldemTrainer::~TexasHoldemTrainer() {
    delete schablone;
}

TexasHoldemTrainer::TexasHoldemTrainer(std::string path) {
    blueprintHandler = nullptr;
    schablone = Template::createDefaultTemplate(path);
}

__global__ void calculatePayoffs(float* dPayoff, float* dPots, bool* dFolded, bool* dPlayer0, int* dNumStateNodes, bool* dPlayerWon, bool* dDraw) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    float* potPointer = dPots + i * 2;
    bool localFolded = dFolded[i];
    bool localPlayer0 = dPlayer0[*dNumStateNodes + i];
    bool currentPlayer = localPlayer0 ? 0 : 1;

    bool localPlayerWon = *dPlayerWon;
    if (localFolded) {
        localPlayerWon = currentPlayer;
    }

    float localPayoff = 0.f;
    if (!*dDraw) {
        localPayoff = potPointer[(currentPlayer + 1) % 2];
    }

    dPayoff[*dNumStateNodes + i] = (localPlayerWon == currentPlayer ? localPayoff : -localPayoff);
}

__global__ void calculateReachProbabilities(float* dReachProbabilities, int* dNumChildren, int* dChildrenWorklistPointers, int* dWorklist, bool* dPlayer0, float* cumulativeRegrets, float* policies, ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    int numChildren = numChildren[i];
    int childrenWorklistPointer = dChildrenWorklistPointers[i];
    int* children = dWorklist + childrenWorklistPointer;
    int currentPlayer = dPlayer0[i] ? 0 : 1;
    int otherPlayer = (currentPlayer + 1) % 2;



    //TODO normalize policies

   /* std::unique_ptr<TrainingInitStruct> trainingInitStruct = initTrainingInitStruct(schablone, i);

    int numChildren = trainingInitStruct->numChildren;
    int* children = trainingInitStruct->children;
    int otherPlayer = trainingInitStruct->otherPlayer;
    int currentPlayer = trainingInitStruct->currentPlayer;
    float* policy = trainingInitStruct->policy;
    float* reachProbabilitiesLocal = trainingInitStruct->reachProbabilitiesLocal;

    for (int j = 0; j < numChildren; j++) {
        if (children[j] < numStateNodes) {
            schablone->structureList->reachProbabilities[2 * children[j] + currentPlayer] = policy[j] * reachProbabilitiesLocal[currentPlayer];
            schablone->structureList->reachProbabilities[2 * children[j] + otherPlayer] = reachProbabilitiesLocal[otherPlayer];
        }
    }
    free(trainingInitStruct->policy);*/
}

//TrainingInitStruct* initTrainingInitStruct(Template* schablone, int i) {
//    int policyPointer = schablone->structureList->policyPointers[i];
//    int numChildren = schablone->structureList->numChildren[i];
//    int childrenWorklistPointer = schablone->structureList->childrenWorklistPointers[i];
//
//    int currentPlayer = schablone->structureList->player0[i] ? 0 : 1;
//    float* cummulativeRegrets = schablone->cumulativeRegrets.at(currentPlayer) + policyPointer;
//    float* policy = (float*)malloc(sizeof(float) * numChildren);
//    std::memcpy(policy, cummulativeRegrets, numChildren * sizeof(float));
//    normalizeStrategy(policy, numChildren);
//    float* reachProbabilitiesLocal = schablone->structureList->reachProbabilities + (i * (size_t)2);
//    int* children = schablone->structureList->worklist + childrenWorklistPointer;
//    int otherPlayer = (currentPlayer + 1) % 2;
//
//    vector<float> reachProbVector;
//    for (int i = 0; i < schablone->structureList->numStateNodes * 2; i++) {
//        reachProbVector.push_back(schablone->structureList->reachProbabilities[i]);
//    }
//
//    TrainingInitStruct trainingInitStruct = TrainingInitStruct();
//
//    trainingInitStruct.policyPointer = policyPointer;
//    trainingInitStruct.numChildren = numChildren;
//    trainingInitStruct.childrenWorklistPointer = childrenWorklistPointer;
//    trainingInitStruct.currentPlayer = currentPlayer;
//    trainingInitStruct.cumulativeRegrets = cummulativeRegrets;
//    trainingInitStruct.policy = policy;
//    trainingInitStruct.reachProbabilitiesLocal = reachProbabilitiesLocal;
//    trainingInitStruct.children = children;
//    trainingInitStruct.otherPlayer = otherPlayer;
//
//    return std::make_unique<TrainingInitStruct>(trainingInitStruct);
//}

int TexasHoldemTrainer::trainSequentiell(int numIterations) {
    Logger::log("Training start");

    int util = 0;
    vector<string> cards;
    cards.reserve(52);
    vector<string> player0Cards;
    player0Cards.reserve(7);
    vector<string> player1Cards;
    player1Cards.reserve(7);
    vector<vector<string>> playerCards = { player0Cards, player1Cards };

    for (int i = 0; i < numIterations; i++) {
        cards = getCards();
        playerCards.at(0) = { cards.at(0), cards.at(1), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8) };
        playerCards.at(1) = { cards.at(2), cards.at(3), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8) };

        if (i % 1000 == 0) {
            std::cout << "train " << i << std::endl;
        }

        util += trainSequentiellIntern(&playerCards);
    }

    saveBucketFunctions();

    Logger::log("Training success");
    return util;
}

void TexasHoldemTrainer::allocateGpuMemory(GpuMemoryPointers* gpuMemoryPointers) {
    int numLeafNodes = schablone->structureList->numLeafNodes;
    int numStateNodes = schablone->structureList->numStateNodes;

    //write
    gpuErrchk(hipMalloc((void**)&gpuMemoryPointers->dPayoff, sizeof(float) * (numLeafNodes + numStateNodes)));
    //TODO
    gpuErrchk(hipMalloc((void**)&gpuMemoryPointers->dPayoff, sizeof(float) * (numLeafNodes + numStateNodes)));

    //read only
    gpuErrchk(hipMalloc((void**)&gpuMemoryPointers->dPots, sizeof(float) * 2 * numLeafNodes));
    gpuErrchk(hipMalloc((void**)&gpuMemoryPointers->dFolded, sizeof(bool) * numLeafNodes));
    gpuErrchk(hipMalloc((void**)&gpuMemoryPointers->dPlayer0, sizeof(bool) * numLeafNodes));
    gpuErrchk(hipMalloc((void**)&gpuMemoryPointers->dNumStateNodes, sizeof(int)));

    //read only, differs in iterations
    gpuErrchk(hipMalloc((void**)&gpuMemoryPointers->dPlayerWon, sizeof(bool)));
    gpuErrchk(hipMalloc((void**)&gpuMemoryPointers->dDraw, sizeof(bool)));

    //TODO
    float reachProbabilitiesLocal[2] = { 1.f, 1.f };
    gpuErrchk(hipMemcpy(gpuMemoryPointers->dReachProbabilities, reachProbabilitiesLocal, sizeof(float) * 2, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(gpuMemoryPointers->dPots, schablone->structureList->pots, sizeof(float) * 2 * numLeafNodes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpuMemoryPointers->dFolded, schablone->structureList->folded, sizeof(bool) * numLeafNodes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpuMemoryPointers->dPlayer0, schablone->structureList->player0, sizeof(bool) * numLeafNodes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(gpuMemoryPointers->dNumStateNodes, &numStateNodes, sizeof(int), hipMemcpyHostToDevice));
}

void TexasHoldemTrainer::cleanUpGpuMemory(GpuMemoryPointers* gpuMemoryPointers) {
    gpuErrchk(hipFree(gpuMemoryPointers->dPayoff));
    gpuErrchk(hipFree(gpuMemoryPointers->dPots));
    gpuErrchk(hipFree(gpuMemoryPointers->dFolded));
    gpuErrchk(hipFree(gpuMemoryPointers->dPlayer0));
    gpuErrchk(hipFree(gpuMemoryPointers->dNumStateNodes));
    gpuErrchk(hipFree(gpuMemoryPointers->dPlayerWon));
    gpuErrchk(hipFree(gpuMemoryPointers->dDraw));
}

int TexasHoldemTrainer::trainGpu(int numIterations) {
    Logger::log("Training start");

    int util = 0;
    vector<string> cards;
    cards.reserve(52);
    vector<string> player0Cards;
    player0Cards.reserve(7);
    vector<string> player1Cards;
    player1Cards.reserve(7);
    vector<vector<string>> playerCards = { player0Cards, player1Cards };

    GpuMemoryPointers* gpuMemoryPointers = new GpuMemoryPointers();

    allocateGpuMemory(gpuMemoryPointers);

    for (int i = 0; i < numIterations; i++) {
        cards = getCards();
        playerCards.at(0) = { cards.at(0), cards.at(1), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8) };
        playerCards.at(1) = { cards.at(2), cards.at(3), cards.at(4), cards.at(5), cards.at(6), cards.at(7), cards.at(8) };

        if (i % 1000 == 0) {
            std::cout << "train " << i << std::endl;
        }

        util += trainGpuIntern(&playerCards, gpuMemoryPointers);
    }

    saveBucketFunctions();

    cleanUpGpuMemory(gpuMemoryPointers);

    Logger::log("Training success");
    return util;
}

void TexasHoldemTrainer::saveBucketFunctions() {
    Logger::log("saveBucketFunctions");
    for (int round = 0; round < 4; round++) {
        schablone->roundInfos.at(round).at(0).bucketFunction->saveBucketFunction();
    }
}

int TexasHoldemTrainer::trainGpuIntern(vector<vector<string>>* playerCards, GpuMemoryPointers* gpuMemoryPointers) {

    //a) bestimme gewinner
    int player0Eval = test7(playerCards->at(0));
    int player1Eval = test7(playerCards->at(1));

    bool draw = player0Eval == player1Eval;
    bool playerWon = player0Eval > player1Eval;

    //b) setze payoffs in leafs durch gewinner
    int numLeafNodes = schablone->structureList->numLeafNodes;
    int numStateNodes = schablone->structureList->numStateNodes;

    hipMemcpy(gpuMemoryPointers->dPlayerWon, &playerWon, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(gpuMemoryPointers->dDraw, &draw, sizeof(bool), hipMemcpyHostToDevice);
    
    dim3 gridSize(1);
    dim3 numLeafNodesBlockSize(numLeafNodes);
    calculatePayoffs<<<gridSize, numLeafNodesBlockSize >>>(gpuMemoryPointers->dPayoff, gpuMemoryPointers->dPots, gpuMemoryPointers->dFolded, gpuMemoryPointers->dPlayer0, gpuMemoryPointers->dNumStateNodes, gpuMemoryPointers->dPlayerWon, gpuMemoryPointers->dDraw);

    gpuErrchk(hipPeekAtLastError());

    //c_1) prepare strategie laden
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            vector<char> bucket = info.bucketFunction->getBucket(playerCards->at(player));
            int pos = info.bucketFunction->getBucketPosition(bucket);
            int size = info.elementSize;

            size_t max = (info.bucketFunction->bucketList.size()) / (info.bucketFunction->size * 2);
            bool newBucket = pos >= max;

            if (newBucket) {
                info.bucketFunction->bucketList.insert(info.bucketFunction->bucketList.end(), bucket.begin(), bucket.end());
                float* zeroArray = new float[size] {0.f};
                info.blueprintHandler->writePolicies(pos, size * sizeof(float), zeroArray);
                delete[](zeroArray);

                int otherPlayer = (player + 1) % 2;
                RoundPlayerInfo otherInfo = schablone->roundInfos.at(round).at(otherPlayer);
                int otherSize = otherInfo.elementSize;
                float* otherZeroArray = new float[otherSize] {0.f};
                otherInfo.blueprintHandler->writePolicies(pos, otherSize * sizeof(float), otherZeroArray);
                delete[](otherZeroArray);
            }
            float* reads = info.blueprintHandler->readPolicies(pos, size * sizeof(float));
            std::memcpy(schablone->cumulativeRegrets.at(player) + info.startPointTemplate, reads, size * sizeof(float));
            delete[] reads;
        }
    }

    //c_2) forwardpass: setze reach probabilities


    for (int i = 0; i < todo; i++) {
        schablone->structureList->levelPointers.at(i) - schablone->structureList->levelPointers.at(i+1);

        

        calculateReachProbabilities << <1, todo >> > ();
    }
    

    

    //TODO cuda barrier

    //d_1) backwardpass: setze regrets
    for (int i = numStateNodes - 1; i >= 0; i--) {
        std::unique_ptr<TrainingInitStruct> trainingInitStruct = initTrainingInitStruct(schablone, i);

        int* children = trainingInitStruct->children;
        int numChildren = trainingInitStruct->numChildren;

        vector<float> upstreamPayoffs;
        upstreamPayoffs.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            upstreamPayoffs.push_back(-1 * schablone->structureList->payoff[children[j]]);
        }

        float* policy = trainingInitStruct->policy;

        float* cumulativeRegrets = trainingInitStruct->cumulativeRegrets;

        float nodeUtility = std::inner_product(policy, policy + numChildren, upstreamPayoffs.begin(), 0.f);
        schablone->structureList->payoff[i] = nodeUtility;

        float* reachProbabilitiesLocal = trainingInitStruct->reachProbabilitiesLocal;
        int currentPlayer = trainingInitStruct->currentPlayer;
        int otherPlayer = trainingInitStruct->otherPlayer;

        float counterValue = reachProbabilitiesLocal[currentPlayer] * nodeUtility;

        vector<float> counterActionValues;
        counterActionValues.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            counterActionValues.push_back(1 * upstreamPayoffs[j]);
        }

        vector<float> counterRegrets;
        counterRegrets.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            counterRegrets.push_back(reachProbabilitiesLocal[otherPlayer] * (counterActionValues[j] - counterValue));

            cumulativeRegrets[j] = cumulativeRegrets[j] + std::max(0.f, counterRegrets[j]);
        }
        free(trainingInitStruct->policy);
    }

    //d_2) postpare strategie zur�ckschreiben
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            int pos = info.bucketFunction->getBucketPosition(info.bucketFunction->getBucket(playerCards->at(player)));

            int size = info.elementSize;

            info.blueprintHandler->writePolicies(pos, size * sizeof(float), schablone->cumulativeRegrets.at(player) + info.startPointTemplate);
        }
    }

    //util?
    return 0;
}

int TexasHoldemTrainer::trainSequentiellIntern(vector<vector<string>>* playerCards) {

    //a) bestimme gewinner
    int player0Eval = test7(playerCards->at(0));
    int player1Eval = test7(playerCards->at(1));

    bool draw = player0Eval == player1Eval;
    bool playerWon = player0Eval > player1Eval;

    //b) setze payoffs in leafs durch gewinner
    int numLeafNodes = schablone->structureList->numLeafNodes;
    int numStateNodes = schablone->structureList->numStateNodes;

    for (int i = 0; i < numLeafNodes; i++) {
        float* potPointer = schablone->structureList->pots + i * 2;
        bool folded = schablone->structureList->folded[i];
        bool player0 = schablone->structureList->player0[numStateNodes + i];
        bool currentPlayer = player0 ? 0 : 1;

        bool localePlayerWon = playerWon;
        if (folded) {
            localePlayerWon = currentPlayer;
        }

        float payoff = 0.f;
        if (!draw) {
            payoff = potPointer[(currentPlayer + 1) % 2];
        }

        schablone->structureList->payoff[numStateNodes + i] = (localePlayerWon == currentPlayer ? payoff : -payoff);
    }

    //c_1) prepare strategie laden
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            vector<char> bucket = info.bucketFunction->getBucket(playerCards->at(player));
            int pos = info.bucketFunction->getBucketPosition(bucket);
            int size = info.elementSize;

            size_t max = (info.bucketFunction->bucketList.size()) / (info.bucketFunction->size * 2);
            bool newBucket = pos >= max;

            if (newBucket) {
                info.bucketFunction->bucketList.insert(info.bucketFunction->bucketList.end(), bucket.begin(), bucket.end());
                float* zeroArray = new float[size] {0.f};
                info.blueprintHandler->writePolicies(pos, size * sizeof(float), zeroArray);
                delete[](zeroArray);

                int otherPlayer = (player + 1) % 2;
                RoundPlayerInfo otherInfo = schablone->roundInfos.at(round).at(otherPlayer);
                int otherSize = otherInfo.elementSize;
                float* otherZeroArray = new float[otherSize] {0.f};
                otherInfo.blueprintHandler->writePolicies(pos, otherSize * sizeof(float), otherZeroArray);
                delete[](otherZeroArray);
            }
            float* reads = info.blueprintHandler->readPolicies(pos, size * sizeof(float));
            std::memcpy(schablone->cumulativeRegrets.at(player) + info.startPointTemplate, reads, size * sizeof(float));
            delete[] reads;
        }
    }

    //c_2) forwardpass: setze reach probabilities
    schablone->structureList->reachProbabilities[0] = 1.f;
    schablone->structureList->reachProbabilities[1] = 1.f;

    for (int i = 0; i < numStateNodes; i++) {
        std::unique_ptr<TrainingInitStruct> trainingInitStruct = initTrainingInitStruct(schablone, i);

        int numChildren = trainingInitStruct->numChildren;
        int* children = trainingInitStruct->children;
        int otherPlayer = trainingInitStruct->otherPlayer;
        int currentPlayer = trainingInitStruct->currentPlayer;
        float* policy = trainingInitStruct->policy;
        float* reachProbabilitiesLocal = trainingInitStruct->reachProbabilitiesLocal;

        for (int j = 0; j < numChildren; j++) {
            if (children[j] < numStateNodes) {
                schablone->structureList->reachProbabilities[2 * children[j] + currentPlayer] = policy[j] * reachProbabilitiesLocal[currentPlayer];
                schablone->structureList->reachProbabilities[2 * children[j] + otherPlayer] = reachProbabilitiesLocal[otherPlayer];
            }
        }
        free(trainingInitStruct->policy);
    }

    //d_1) backwardpass: setze regrets
    for (int i = numStateNodes - 1; i >= 0; i--) {
        std::unique_ptr<TrainingInitStruct> trainingInitStruct = initTrainingInitStruct(schablone, i);

        int* children = trainingInitStruct->children;
        int numChildren = trainingInitStruct->numChildren;

        vector<float> upstreamPayoffs;
        upstreamPayoffs.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            upstreamPayoffs.push_back(-1 * schablone->structureList->payoff[children[j]]);
        }

        float* policy = trainingInitStruct->policy;

        float* cumulativeRegrets = trainingInitStruct->cumulativeRegrets;

        float nodeUtility = std::inner_product(policy, policy + numChildren, upstreamPayoffs.begin(), 0.f);
        schablone->structureList->payoff[i] = nodeUtility;

        float* reachProbabilitiesLocal = trainingInitStruct->reachProbabilitiesLocal;
        int currentPlayer = trainingInitStruct->currentPlayer;
        int otherPlayer = trainingInitStruct->otherPlayer;

        float counterValue = reachProbabilitiesLocal[currentPlayer] * nodeUtility;

        vector<float> counterActionValues;
        counterActionValues.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            counterActionValues.push_back(1 * upstreamPayoffs[j]);
        }

        vector<float> counterRegrets;
        counterRegrets.reserve(numChildren);

        for (int j = 0; j < numChildren; j++) {
            counterRegrets.push_back(reachProbabilitiesLocal[otherPlayer] * (counterActionValues[j] - counterValue));

            cumulativeRegrets[j] = cumulativeRegrets[j] + std::max(0.f, counterRegrets[j]);
        }
        free(trainingInitStruct->policy);
    }

    //d_2) postpare strategie zur�ckschreiben
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            RoundPlayerInfo info = schablone->roundInfos.at(round).at(player);
            int pos = info.bucketFunction->getBucketPosition(info.bucketFunction->getBucket(playerCards->at(player)));

            int size = info.elementSize;

            info.blueprintHandler->writePolicies(pos, size * sizeof(float), schablone->cumulativeRegrets.at(player) + info.startPointTemplate);
        }
    }

    //util?
    return 0;
}