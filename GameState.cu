#include "hip/hip_runtime.h"
#include "GameState.cuh"
#include "StateNode.cuh"
#include "LeafNode.cuh"
#include "Template.cuh"
#include "Utils.cuh"
#include "RaiseBuckets.cuh"

#include <algorithm>
#include <iostream>

GameState::GameState() {
    player0 = true;
    history = {};
    round = 0;
    pot = { 0,0 };
}

GameState::GameState(const GameState& gameState) {
    player0 = gameState.player0;
    history = gameState.history;
    round = gameState.round;
    pot = gameState.pot;
}

vector<pair<char, float>> GameState::getActions() {
    vector<pair<char, float>> actions{ pair<char,float>('c',0.f) };

    if ((history.size() != 0 && history.back() == 'r')) actions.push_back(pair<char, float>('f', 0.f));
    if (history.size() < 2) {
        vector<pair<char, float>> raiseSizes = getRaises();
        for (pair<char, float> raiseSize : raiseSizes) {
            actions.push_back(raiseSize);
        }
    }

    return actions;
}

/**
 * Spieler ist der alte(?)
 */
void GameState::adjustPot(pair<char, float> action) {
    if (action.first == 'f') return;

    if (action.first == 'c') {
        player0 ? pot.first = pot.second : pot.second = pot.first;
        return;
    }

    if (action.first == 'r') {
        player0 ? pot.first = pot.second += action.second : pot.second = pot.first + action.second;
        return;
    }
}

/**
 * action + gamestate -> gibt Knoten und neuen GameState
 * Knoten kann State/LEaf
 * Leaf -> pot und durch isFold
 * State -> -
 */
struct HandleActionReturnType GameState::handleAction(pair<char, float> action) {

    GameState thisGameState = *this;
    GameState* gameState = new GameState(thisGameState);

    gameState->adjustPot(action);

    if (action.first == 'f') {
        gameState->player0 = !gameState->player0;
        LeafNode* node = new LeafNode();
        node->pot = gameState->pot;
        node->folded = true;
        return HandleActionReturnType { node, false, gameState };

    }
    else if (roundEnd(history, action.first)) {
        if (round == 3) {
            gameState->player0 = !gameState->player0;
            LeafNode* node = new LeafNode();
            node->pot = gameState->pot;
            node->folded = false;
            return HandleActionReturnType { node, false, gameState };

        }
        else {
            gameState->player0 = true;
            gameState->history = {};
            gameState->round++;
            StateNode* node = new StateNode();
            return HandleActionReturnType { node, true, gameState };
        }

    }
    else {
        gameState->player0 = !gameState->player0;
        gameState->history.push_back(action.first);
        StateNode* node = new StateNode();
        return HandleActionReturnType { node, true, gameState };
    }
}