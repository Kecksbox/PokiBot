#include "hip/hip_runtime.h"
#include "Template.cuh"
#include "GameState.cuh"
#include "StateNode.cuh"
#include "LeafNode.cuh"

#include <iostream>
#include <algorithm>
#include <cstring>

Template::~Template() {
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            delete roundInfos.at(round).at(player).blueprintHandler;
        }
        delete roundInfos.at(round).at(0).bucketFunction;
    }

    delete structureList;

}

Template::Template(StructureList* structureList, vector<vector<RoundPlayerInfo>> roundInfos, vector<vector<float>> cumulativeRegrets) {
    this->roundInfos = roundInfos;
    this->structureList = structureList;
    this->cumulativeRegrets = cumulativeRegrets;
}

Template::Template(Template* schablone) {
    this->cumulativeRegrets = schablone->cumulativeRegrets;
    this->roundInfos = schablone->roundInfos;
    this->structureList = schablone->structureList;
}

void Template::createBucketFunctions(std::string path, vector<BucketFunction*>* bucketFunctions) {
    BucketFunction* bucketFunction0 = new BucketFunction(path + "_buckets_" + "0", 0, 2);
    bucketFunctions->push_back(bucketFunction0);
    BucketFunction* bucketFunction1 = new BucketFunction(path + "_buckets_" + "1", 1, 5);
    bucketFunctions->push_back(bucketFunction1);
    BucketFunction* bucketFunction2 = new BucketFunction(path + "_buckets_" + "2", 2, 6);
    bucketFunctions->push_back(bucketFunction2);
    BucketFunction* bucketFunction3 = new BucketFunction(path + "_buckets_" + "3", 3, 7);
    bucketFunctions->push_back(bucketFunction3);
}

struct BuildTreeReturnType Template::buildTree() {
    vector<vector<int>> roundPlayerActionCounts(4, vector<int>(2, 0));

    StateNode* root = new StateNode();
    GameState* gameState = new GameState();

    vector<struct NodeInformation> nodeInformations;
    vector<LeafNode*> leafNodes;
    vector<StateNode*> stateNodes;

    stateNodes.push_back(root);

    struct NodeInformation nodeInformation = { 0, true, gameState };
    nodeInformations.push_back(nodeInformation);

    int index = 0;
    while (index < nodeInformations.size()) {

        struct NodeInformation currentNodeInformation = nodeInformations.at(index);
        int currentNodeIndex = currentNodeInformation.nodeIndex;
        bool currentNodeIsStateNode = currentNodeInformation.isStateNode;
        GameState* currentGameState = currentNodeInformation.gameState;

        index++;
        Node* currentNode;

        if (currentNodeIsStateNode) {
            currentNode = stateNodes.at(currentNodeIndex);

        }
        else {
            currentNode = leafNodes.at(currentNodeIndex);
        }
        currentNode->player0 = currentGameState->player0;
        currentNode->payoff = 0.f;

        if (currentNodeIsStateNode) {
            StateNode* currentStateNode = (StateNode*)currentNode;
            vector<pair<char, float>> actions = currentGameState->getActions();

            roundPlayerActionCounts[currentGameState->round][currentGameState->player0 ? 0 : 1] += actions.size();

            for (pair<char, float> action : actions) {
                HandleActionReturnType handleActionReturnType = currentGameState->handleAction(action);
                int nodeInformationsSize = nodeInformations.size();

                int nodeIndex = 0;
                if (handleActionReturnType.isStateNode) {
                    nodeIndex = stateNodes.size();
                    stateNodes.push_back((StateNode*)handleActionReturnType.node);
                }
                else {
                    nodeIndex = leafNodes.size();
                    leafNodes.push_back(((LeafNode*)handleActionReturnType.node));
                }

                nodeInformations.push_back({ nodeIndex, handleActionReturnType.isStateNode, handleActionReturnType.gameState });
                currentStateNode->children.push_back(nodeInformationsSize);
            }
        }
    }

    int stateNodesSize = stateNodes.size();
    int leafNodesSize = leafNodes.size();
    int worklistSize = stateNodesSize + leafNodesSize;
    auto stateWorklist = vector<StateNode>(stateNodesSize);;
    auto leafWorklist = vector<LeafNode>(leafNodesSize);

    auto worklist = vector<int>(worklistSize);

    for (int i = 0; i < worklistSize; i++) {
        nodeInformation = nodeInformations.at(i);
        int nodeIndex = nodeInformation.nodeIndex;
        if (nodeInformation.isStateNode) {
            stateWorklist[nodeIndex] = *stateNodes.at(nodeIndex);
        }
        else {
            leafWorklist[nodeIndex] = *leafNodes.at(nodeIndex);
            nodeIndex += stateNodesSize;
        }
        worklist[i] = nodeIndex;
    }

    //cleanup GameStates + Nodes
    for (int i = 0; i < worklistSize; i++) {
        delete nodeInformations.at(i).gameState;

        nodeInformation = nodeInformations.at(i);
        int nodeIndex = nodeInformation.nodeIndex;
        if (nodeInformation.isStateNode) {
            delete stateNodes.at(nodeIndex);
        }
        else {
            delete leafNodes.at(nodeIndex);
        }
    }

    struct BuildTreeReturnType buildTreeReturnType = { worklist, worklistSize, stateWorklist, stateNodesSize, leafWorklist, roundPlayerActionCounts };
    return buildTreeReturnType;
}

static vector<vector<RoundPlayerInfo>> buildRoundPlayerInfos(vector<BucketFunction*>* bucketFunctions, vector<vector<int>>* roundPlayerActionCounts) {
    vector<vector<RoundPlayerInfo>> roundPlayerInfos;

    int templatePointers[2] = { 0,0 };

    for (int round = 0; round < 4; round++) {
        vector<RoundPlayerInfo> temp;
        roundPlayerInfos.push_back(temp);
        for (int player = 0; player < 2; player++) {
            BucketFunction* bucketFunction = bucketFunctions->at(round);
            int elementSize = roundPlayerActionCounts->at(round).at(player);
            int startPointTemplate = templatePointers[player];
            templatePointers[player] += elementSize;

            RoundPlayerInfo roundPlayerInfo = RoundPlayerInfo(startPointTemplate, elementSize, bucketFunction, round, player);
            roundPlayerInfos.at(round).push_back(roundPlayerInfo);
        }
    }
    return roundPlayerInfos;
}

static void reduceRoundPlayerActionCounts(vector<vector<int>>* roundPlayerActionCounts, int playerActionCounts[2]) {
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            playerActionCounts[player] += roundPlayerActionCounts->at(round).at(player);
        }
    }
}

//backwardPass durch worklist
static void worklistBackwardPass(vector<vector<float>>* cumulativeRegrets, vector<int>* worklist, int worklistLength, vector<StateNode>* stateWorklist, int stateWorklistLength, vector<vector<int>>* roundPlayerActionCounts) {
    int playerActionCounts[2] = { 0,0 };
    reduceRoundPlayerActionCounts(roundPlayerActionCounts, playerActionCounts);

    vector<float> player0CumulativeRegrets = vector<float>(playerActionCounts[0]);
    vector<float> player1CumulativeRegrets = vector<float>(playerActionCounts[1]);

    cumulativeRegrets->push_back(player0CumulativeRegrets);
    cumulativeRegrets->push_back(player1CumulativeRegrets);

    int templatePointers[2] = { 0, 0 };

    for (int i = worklistLength - 1; i >= 0; i--) {
        int worklistPointer = worklist->at(i);
        if (worklistPointer < stateWorklistLength) {
            StateNode* stateNode = &stateWorklist->at(worklistPointer);
            stateNode->policyPointer = playerActionCounts[stateNode->player0 ? 0 : 1] - (templatePointers[stateNode->player0 ? 0 : 1] + stateNode->children.size());
            templatePointers[stateNode->player0 ? 0 : 1] += stateNode->children.size();
        }
    }
}

StructureList* treeToLists(struct BuildTreeReturnType* tree) {
    int numStateNodes = tree->stateWorklistLength;
    int numLeafNodes = tree->worklistLength - numStateNodes;
    int numNodes = numStateNodes + numLeafNodes;

    std::vector<int> worklist = tree->worklist;
    auto payoff = vector<float>(numNodes);
    auto player0 = vector<bool>(numNodes);

    auto numChildren = vector<int>(numStateNodes);
    auto policyPointers = vector<int>(numStateNodes);
    auto childrenWorklistPointers = vector<int>(numStateNodes);
    auto reachProbabilities = vector<float>(2 * numStateNodes);

    auto pots = vector<float>(2 * numLeafNodes);
    auto folded = vector<bool>(numLeafNodes);

    for (int i = 0; i < numStateNodes; i++) {
        StateNode* stateNode = &tree->stateWorklist[i];
        // GLOBALE DATEN
        player0[i] = stateNode->player0;
        //TODO stateNodes brauchen keinen payoff mehr, dieser wird eh �berschrieben w�hrend des Trainings.
        // STATE NODE DATEN
        numChildren[i] = stateNode->children.size();
        policyPointers[i] = stateNode->policyPointer;

        vector<int> children = stateNode->children;
        std::sort(children.begin(), children.end());

        if ((size_t)children.back() - (size_t)children.at(0) != children.size() - (size_t)1) {
            throw "Assertion failed, empty space in children vector found";
        }

        childrenWorklistPointers[i] = children.at(0);

    }

    for (int i = 0; i < numLeafNodes; i++) {
        LeafNode* leafNode = &tree->leafWorklist[i];
        player0[i + numStateNodes] = leafNode->player0;

        pots[i * 2] = leafNode->pot.first;
        pots[(i * 2) + 1] = leafNode->pot.second;

        folded[i] = leafNode->folded;

    }

    //Ebeneninformationen
    vector<int> levelPointers = { 0 };
    int pointer = 0;
    for (int i = 0; i < numStateNodes; i++) {
        int startingLocalMinChildIndex = childrenWorklistPointers[i];
        int localMinChildIndexNumChildren = numChildren[i];
        int localMinChildIndex = numNodes + 1;
        for (int j = 0; j < localMinChildIndexNumChildren; j++) {
            if (worklist[startingLocalMinChildIndex + j] < numStateNodes) {
                localMinChildIndex = std::min(localMinChildIndex, startingLocalMinChildIndex + j);
            }
        }

        if (levelPointers.at(pointer) == i) {
            pointer++;
            levelPointers.push_back(localMinChildIndex);
        }
        else {
            levelPointers.at(pointer) = std::min(localMinChildIndex, levelPointers.at(pointer));
        }
    }

    //TODO wie kann man den Konstruktor direkt mit Parametern aufrufen?
    StructureList* structureList = new StructureList();
    structureList->childrenWorklistPointers = childrenWorklistPointers;
    structureList->folded = folded;
    structureList->levelPointers = levelPointers;
    structureList->numChildren = numChildren;
    structureList->numStateNodes = numStateNodes;
    structureList->numLeafNodes = numLeafNodes;
    structureList->payoff = payoff;
    structureList->player0 = player0;
    structureList->policyPointers = policyPointers;
    structureList->pots = pots;
    structureList->reachProbabilities = reachProbabilities;
    structureList->worklist = worklist;
    return structureList;
}

Template* Template::createDefaultTemplate(std::string path) {
    vector<BucketFunction*> bucketFunctions;
    createBucketFunctions(path, &bucketFunctions);

    struct BuildTreeReturnType tree = buildTree();
    vector<vector<RoundPlayerInfo>> roundPlayerInfos = buildRoundPlayerInfos(&bucketFunctions, &tree.roundPlayerActionCounts);

    vector<vector<float>> cumulativeRegrets;

    worklistBackwardPass(&cumulativeRegrets, &tree.worklist, tree.worklistLength, &tree.stateWorklist, tree.stateWorklistLength, &tree.roundPlayerActionCounts);

    StructureList* listCollection = treeToLists(&tree);

    return new Template(std::move(listCollection), roundPlayerInfos, cumulativeRegrets);
}