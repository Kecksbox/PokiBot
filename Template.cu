#include "hip/hip_runtime.h"
#include "Template.cuh"
#include "GameState.cuh"
#include "StateNode.cuh"
#include "LeafNode.cuh"

#include <iostream>
#include <algorithm>
#include <cstring>

Template::~Template() {
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            delete roundInfos.at(round).at(player).blueprintHandler;
        }
        delete roundInfos.at(round).at(0).bucketFunction;
    }

    delete[] structureList->payoff;
    delete[] structureList->player0;
    delete[] structureList->numChildren;
    delete[] structureList->policyPointers;
    delete[] structureList->childrenWorklistPointers;
    delete[] structureList->reachProbabilities;
    delete[] structureList->pots;
    delete[] structureList->folded;

    free(structureList->worklist);

    delete structureList;

    for (int i = 0; i < cumulativeRegrets.size(); i++) {
        free(cumulativeRegrets.at(i));
    }
}

Template::Template(StructureList* structureList, vector<vector<RoundPlayerInfo>> roundInfos, vector<float*> cumulativeRegrets) {
    this->roundInfos = roundInfos;
    this->structureList = structureList;
    this->cumulativeRegrets = cumulativeRegrets;
}

Template::Template(Template* schablone) {
    this->cumulativeRegrets = schablone->cumulativeRegrets;
    this->roundInfos = schablone->roundInfos;
    this->structureList = schablone->structureList;
}

void Template::createBucketFunctions(std::string folder, std::string fileName, vector<BucketFunction*>* bucketFunctions) {
    BucketFunction* bucketFunction0 = new BucketFunction(folder, fileName + "_buckets_" + "0", 0, 2);
    bucketFunctions->push_back(bucketFunction0);
    BucketFunction* bucketFunction1 = new BucketFunction(folder, fileName + "_buckets_" + "1", 1, 5);
    bucketFunctions->push_back(bucketFunction1);
    BucketFunction* bucketFunction2 = new BucketFunction(folder, fileName + "_buckets_" + "2", 2, 6);
    bucketFunctions->push_back(bucketFunction2);
    BucketFunction* bucketFunction3 = new BucketFunction(folder, fileName + "_buckets_" + "3", 3, 7);
    bucketFunctions->push_back(bucketFunction3);
}

struct BuildTreeReturnType Template::buildTree() {
    vector<vector<int>> roundPlayerActionCounts(4, vector<int>(2, 0));

    StateNode* root = new StateNode();
    GameState* gameState = new GameState();

    vector<struct NodeInformation> nodeInformations;
    vector<LeafNode*> leafNodes;
    vector<StateNode*> stateNodes;

    stateNodes.push_back(root);

    struct NodeInformation nodeInformation = { 0, true, gameState };
    nodeInformations.push_back(nodeInformation);

    int index = 0;
    while (index < nodeInformations.size()) {

        struct NodeInformation currentNodeInformation = nodeInformations.at(index);
        int currentNodeIndex = currentNodeInformation.nodeIndex;
        bool currentNodeIsStateNode = currentNodeInformation.isStateNode;
        GameState* currentGameState = currentNodeInformation.gameState;

        index++;
        Node* currentNode;

        if (currentNodeIsStateNode) {
            currentNode = stateNodes.at(currentNodeIndex);

        }
        else {
            currentNode = leafNodes.at(currentNodeIndex);
        }
        currentNode->player0 = currentGameState->player0;
        currentNode->payoff = 0.f;

        if (currentNodeIsStateNode) {
            StateNode* currentStateNode = (StateNode*)currentNode;
            vector<pair<char, float>> actions = currentGameState->getActions();

            roundPlayerActionCounts[currentGameState->round][currentGameState->player0 ? 0 : 1] += actions.size();

            for (pair<char, float> action : actions) {
                HandleActionReturnType handleActionReturnType = currentGameState->handleAction(action);
                int nodeInformationsSize = nodeInformations.size();

                int nodeIndex = 0;
                if (handleActionReturnType.isStateNode) {
                    nodeIndex = stateNodes.size();
                    stateNodes.push_back((StateNode*)handleActionReturnType.node);
                }
                else {
                    nodeIndex = leafNodes.size();
                    leafNodes.push_back(((LeafNode*)handleActionReturnType.node));
                }

                nodeInformations.push_back({ nodeIndex, handleActionReturnType.isStateNode, handleActionReturnType.gameState });
                currentStateNode->children.push_back(nodeInformationsSize);
            }
        }
    }

    int stateNodesSize = stateNodes.size();
    int leafNodesSize = leafNodes.size();
    int worklistSize = stateNodesSize + leafNodesSize;
    auto stateWorklist = new StateNode[stateNodesSize];
    LeafNode* leafWorklist = new LeafNode[leafNodesSize];

    int* worklist = (int*)malloc((worklistSize) * sizeof(int));

    for (int i = 0; i < worklistSize; i++) {
        nodeInformation = nodeInformations.at(i);
        int nodeIndex = nodeInformation.nodeIndex;
        if (nodeInformation.isStateNode) {
            stateWorklist[nodeIndex] = *stateNodes.at(nodeIndex);
        }
        else {
            leafWorklist[nodeIndex] = *leafNodes.at(nodeIndex);
            nodeIndex += stateNodesSize;
        }
        worklist[i] = nodeIndex;
    }

    for (int i = 0; i < worklistSize; i++) {
        delete nodeInformations.at(i).gameState;

        nodeInformation = nodeInformations.at(i);
        int nodeIndex = nodeInformation.nodeIndex;
        if (nodeInformation.isStateNode) {
            delete stateNodes.at(nodeIndex);
        }
        else {
            delete leafNodes.at(nodeIndex);
        }
    }

    struct BuildTreeReturnType buildTreeReturnType = { worklist, worklistSize, stateWorklist, stateNodesSize, leafWorklist, roundPlayerActionCounts };
    return buildTreeReturnType;
}

static vector<vector<RoundPlayerInfo>> buildRoundPlayerInfos(std::string folder, std::string fileName, vector<BucketFunction*>* bucketFunctions, vector<vector<int>>* roundPlayerActionCounts) {
    vector<vector<RoundPlayerInfo>> roundPlayerInfos;

    int templatePointers[2] = { 0,0 };

    for (int round = 0; round < 4; round++) {
        vector<RoundPlayerInfo> temp;
        roundPlayerInfos.push_back(temp);
        for (int player = 0; player < 2; player++) {
            BucketFunction* bucketFunction = bucketFunctions->at(round);
            int elementSize = roundPlayerActionCounts->at(round).at(player);
            int startPointTemplate = templatePointers[player];
            templatePointers[player] += elementSize;

            RoundPlayerInfo roundPlayerInfo = RoundPlayerInfo(folder, fileName, startPointTemplate, elementSize, bucketFunction, round, player);
            roundPlayerInfos.at(round).push_back(roundPlayerInfo);
        }
    }
    return roundPlayerInfos;
}

static void reduceRoundPlayerActionCounts(vector<vector<int>>* roundPlayerActionCounts, int playerActionCounts[2]) {
    for (int round = 0; round < 4; round++) {
        for (int player = 0; player < 2; player++) {
            playerActionCounts[player] += roundPlayerActionCounts->at(round).at(player);
        }
    }
}

static void worklistBackwardPass(vector<float*>* cumulativeRegrets, int* worklist, int worklistLength, StateNode* stateWorklist, int stateWorklistLength, vector<vector<int>>* roundPlayerActionCounts) {
    int playerActionCounts[2] = { 0,0 };
    reduceRoundPlayerActionCounts(roundPlayerActionCounts, playerActionCounts);

    float* player0CumulativeRegrets = (float*)malloc(playerActionCounts[0] * sizeof(float));
    float* player1CumulativeRegrets = (float*)malloc(playerActionCounts[1] * sizeof(float));

    cumulativeRegrets->push_back(player0CumulativeRegrets);
    cumulativeRegrets->push_back(player1CumulativeRegrets);

    int templatePointers[2] = { 0, 0 };

    for (int i = worklistLength - 1; i >= 0; i--) {
        int worklistPointer = worklist[i];
        if (worklistPointer < stateWorklistLength) {
            StateNode* stateNode = &stateWorklist[worklistPointer];
            stateNode->policyPointer = playerActionCounts[stateNode->player0 ? 0 : 1] - (templatePointers[stateNode->player0 ? 0 : 1] + stateNode->children.size());
            templatePointers[stateNode->player0 ? 0 : 1] += stateNode->children.size();
        }
    }
}

StructureList* treeToLists(struct BuildTreeReturnType* tree) {
    int numStateNodes = tree->stateWorklistLength;
    int numLeafNodes = tree->worklistLength - numStateNodes;
    int numNodes = numStateNodes + numLeafNodes;

    int* worklist = tree->worklist;
    float* payoff = new float[numNodes];
    bool* player0 = new bool[numNodes];

    int* numChildren = new int[numStateNodes];
    int* policyPointers = new int[numStateNodes];
    int* childrenWorklistPointers = new int[numStateNodes];
    float* reachProbabilities = new float[(size_t)2 * numStateNodes];

    float* pots = new float[(size_t)2 * numLeafNodes];
    bool* folded = new bool[numLeafNodes];

    for (int i = 0; i < numStateNodes; i++) {
        StateNode* stateNode = &tree->stateWorklist[i];
        player0[i] = stateNode->player0;
        numChildren[i] = stateNode->children.size();
        policyPointers[i] = stateNode->policyPointer;

        vector<int> children = stateNode->children;
        std::sort(children.begin(), children.end());

        if ((size_t)children.back() - (size_t)children.at(0) != children.size() - (size_t)1) {
            throw "Assertion failed, empty space in children vector found";
        }

        childrenWorklistPointers[i] = children.at(0);

    }

    for (int i = 0; i < numLeafNodes; i++) {
        LeafNode* leafNode = &tree->leafWorklist[i];
        player0[i + numStateNodes] = leafNode->player0;

        pots[i * 2] = leafNode->pot.first;
        pots[(i * 2) + 1] = leafNode->pot.second;

        folded[i] = leafNode->folded;

    }

    vector<int> levelPointers = { 0 };

    int minEbene = numNodes + 1;

    for (int i = 0; i < numNodes; i++) {

        if (worklist[i] < numStateNodes) {

            int minChildOfI = numNodes + 1;
            for (int j = 0; j < numChildren[worklist[i]]; j++) {
                if (worklist[childrenWorklistPointers[worklist[i]] + j] < numStateNodes) {
                    minChildOfI = std::min(minChildOfI, childrenWorklistPointers[worklist[i]] + j);
                }

                minEbene = std::min(minEbene, minChildOfI);
            }
        }

        if (i + 1 == minEbene) {
            levelPointers.push_back(minEbene);
            minEbene = numNodes + 1;
        }
    }

    delete[] tree->stateWorklist;
    delete[] tree->leafWorklist;

    StructureList* structureList = new StructureList();
    structureList->childrenWorklistPointers = childrenWorklistPointers;
    structureList->folded = folded;
    structureList->levelPointers = levelPointers;
    structureList->numChildren = numChildren;
    structureList->numStateNodes = numStateNodes;
    structureList->numLeafNodes = numLeafNodes;
    structureList->payoff = payoff;
    structureList->player0 = player0;
    structureList->policyPointers = policyPointers;
    structureList->pots = pots;
    structureList->reachProbabilities = reachProbabilities;
    structureList->worklist = worklist;
    return structureList;
}

Template* Template::createDefaultTemplate(std::string folder, std::string fileName) {
    vector<BucketFunction*> bucketFunctions;
    createBucketFunctions(folder, fileName, &bucketFunctions);

    struct BuildTreeReturnType tree = buildTree();
    vector<vector<RoundPlayerInfo>> roundPlayerInfos = buildRoundPlayerInfos(folder, fileName, &bucketFunctions, &tree.roundPlayerActionCounts);

    vector<float*> cumulativeRegrets;

    worklistBackwardPass(&cumulativeRegrets, tree.worklist, tree.worklistLength, tree.stateWorklist, tree.stateWorklistLength, &tree.roundPlayerActionCounts);

    StructureList* listCollection = treeToLists(&tree);

    return new Template(std::move(listCollection), roundPlayerInfos, cumulativeRegrets);
}