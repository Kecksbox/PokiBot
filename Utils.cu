#include "hip/hip_runtime.h"
#include "Utils.cuh"
#include "Cards.cuh"
#include <cstring>
#include "RaiseBuckets.cuh"

// std::default_random_engine engine = std::default_random_engine();
std::default_random_engine engine;

void normalizeStrategy(std::vector<float> policy, int size) {
    for (int i = 0; i < size; i++) {
        policy.at(i) = std::max(policy.at(i), 0.f);
    }

    float arraySum = 0;

    for (int i = 0; i < size; i++) {
        arraySum += policy.at(i);
    }

    if (arraySum > 0) {
        for (int i = 0; i < size; i++) {
            policy.at(i) /= arraySum;
        }
    }
    else {
        for (int i = 0; i < size; i++) {
            policy.at(i) = 1.f / size;
        }
    }
}

std::unique_ptr<TrainingInitStruct> initTrainingInitStruct(Template* schablone, int i) {
    int policyPointer = schablone->structureList->policyPointers.at(i);
    int numChildren = schablone->structureList->numChildren.at(i);
    int childrenWorklistPointer = schablone->structureList->childrenWorklistPointers.at(i);

    int currentPlayer = schablone->structureList->player0.at(i) ? 0 : 1;
	std::vector<float> cummulativeRegrets(schablone->cumulativeRegrets.at(currentPlayer).begin() + policyPointer, schablone->cumulativeRegrets.at(currentPlayer).begin() + policyPointer + numChildren);
    std::vector<float> policy(cummulativeRegrets);
    normalizeStrategy(policy, numChildren);
	//float* reachProbabilitiesLocal = schablone->structureList->reachProbabilities + (i* (size_t)2);
    std::vector<float> reachProbabilitiesLocal(schablone->structureList->reachProbabilities.begin() + i * 2, schablone->structureList->reachProbabilities.begin() + i * 2 + 2);
	//int* children = schablone->structureList->worklist + childrenWorklistPointer;
    std::vector<int> children(schablone->structureList->worklist.begin() + childrenWorklistPointer, schablone->structureList->worklist.begin() + childrenWorklistPointer + numChildren);
    int otherPlayer = (currentPlayer + 1) % 2;

    vector<float> reachProbVector;
    for (int i = 0; i < schablone->structureList->numStateNodes * 2; i++) {
        reachProbVector.push_back(schablone->structureList->reachProbabilities.at(i));
    }

    TrainingInitStruct trainingInitStruct = TrainingInitStruct();

    trainingInitStruct.policyPointer = policyPointer;
    trainingInitStruct.numChildren = numChildren;
    trainingInitStruct.childrenWorklistPointer = childrenWorklistPointer;
    trainingInitStruct.currentPlayer = currentPlayer;
    trainingInitStruct.cumulativeRegrets = cummulativeRegrets;
    trainingInitStruct.policy = policy;
    trainingInitStruct.reachProbabilitiesLocal = reachProbabilitiesLocal;
    trainingInitStruct.children = children;
    trainingInitStruct.otherPlayer = otherPlayer;

    return std::make_unique<TrainingInitStruct>(trainingInitStruct);
}

bool roundEnd(vector<char> history, char action) {
    return !history.empty() && ((history.back() == 'c' || history.back() == 'r') && action == 'c');
}

bool roundEnd(vector<pair<char, float>> history, pair<char, float> action) {
    return !history.empty() && ((history.back().first == 'c' || history.back().first == 'r') && action.first == 'c');
}

vector<string> getCards() {
    vector<std::string> cards = Cards::getCards();
    std::shuffle(cards.begin(), cards.end(), engine);

    return cards;
}

vector<string> mapCardsToVisibility(vector<string> cards, int player, int round) {
    vector<string> visibleCards;

    visibleCards.push_back(player == 0 ? cards.at(0) : cards.at(2));
    visibleCards.push_back(player == 0 ? cards.at(1) : cards.at(3));

    if (round >= 1) {
        visibleCards.push_back(cards.at(4));
        visibleCards.push_back(cards.at(5));
        visibleCards.push_back(cards.at(6));
        if (round >= 2) {
            visibleCards.push_back(cards.at(7));
            if (round >= 3) {
                visibleCards.push_back(cards.at(8));
            }
        }
    }

    return visibleCards;
}

std::pair<int, GameState*> getCurrentNode(Template* schablone, std::vector<std::pair<char, float>> actionHistory) {

    // init with root
    int currentNode = schablone->structureList->worklist.at(0);
    GameState* currentGameState = new GameState();

    for (int i = 0; i < actionHistory.size(); i++) {

        pair<char, float> currentAction = actionHistory.at(i).first == 'r' ? pair<char, float>('r', getRaise(actionHistory.at(i).second)) : actionHistory.at(i);

        vector<pair<char, float>> possibleActions = currentGameState->getActions();

        std::vector<pair<char, float>>::iterator it = std::find(possibleActions.begin(), possibleActions.end(), currentAction);

        int index = it - possibleActions.begin();

        int currentNodeIndex = schablone->structureList->childrenWorklistPointers.at(currentNode) + index;

        currentNode = schablone->structureList->worklist[currentNodeIndex];

        currentGameState = currentGameState->handleAction(currentAction).gameState;

    }

    return pair<int, GameState*>(currentNode, currentGameState);

}