#include "hip/hip_runtime.h"
#include "Utils.cuh"
#include "Cards.cuh"
#include <cstring>
#include "RaiseBuckets.cuh"

// std::default_random_engine engine = std::default_random_engine();
std::default_random_engine engine;

void normalizeStrategy(float* policy, int size) {
    for (int i = 0; i < size; i++) {
        policy[i] = std::max(policy[i], 0.f);
    }

    float arraySum = 0;

    for (int i = 0; i < size; i++) {
        arraySum += policy[i];
    }

    if (arraySum > 0) {
        for (int i = 0; i < size; i++) {
            policy[i] /= arraySum;
        }
    }
    else {
        for (int i = 0; i < size; i++) {
            policy[i] = 1.f / size;
        }
    }
}

TrainingInitStruct* initTrainingInitStruct(Template* schablone, int i) {
    int policyPointer = schablone->structureList->policyPointers[i];
    int numChildren = schablone->structureList->numChildren[i];
    int childrenWorklistPointer = schablone->structureList->childrenWorklistPointers[i];

    int currentPlayer = schablone->structureList->player0[i] ? 0 : 1;
    float* cummulativeRegrets = schablone->cumulativeRegrets.at(currentPlayer) + policyPointer;
    float* policy = (float*)malloc(sizeof(float) * numChildren);
    std::memcpy(policy, cummulativeRegrets, numChildren * sizeof(float));
    normalizeStrategy(policy, numChildren);
    float* reachProbabilitiesLocal = schablone->structureList->reachProbabilities + (i * (size_t)2);
    int* children = schablone->structureList->worklist + childrenWorklistPointer;
    int otherPlayer = (currentPlayer + 1) % 2;

    auto trainingInitStruct = new TrainingInitStruct();

    trainingInitStruct->policyPointer = policyPointer;
    trainingInitStruct->numChildren = numChildren;
    trainingInitStruct->childrenWorklistPointer = childrenWorklistPointer;
    trainingInitStruct->currentPlayer = currentPlayer;
    trainingInitStruct->cumulativeRegrets = cummulativeRegrets;
    trainingInitStruct->policy = policy;
    trainingInitStruct->reachProbabilitiesLocal = reachProbabilitiesLocal;
    trainingInitStruct->children = children;
    trainingInitStruct->otherPlayer = otherPlayer;

    return trainingInitStruct;
}

bool roundEnd(vector<char> history, char action) {
    return !history.empty() && ((history.back() == 'c' || history.back() == 'r') && action == 'c');
}

bool roundEnd(vector<pair<char, float>> history, pair<char, float> action) {
    return !history.empty() && ((history.back().first == 'c' || history.back().first == 'r') && action.first == 'c');
}

vector<string> getCards() {
    vector<std::string> cards = Cards::getCards();
    std::shuffle(cards.begin(), cards.end(), engine);

    return cards;
}

vector<string> mapCardsToVisibility(vector<string> cards, int player, int round) {
    vector<string> visibleCards;

    visibleCards.push_back(player == 0 ? cards.at(0) : cards.at(2));
    visibleCards.push_back(player == 0 ? cards.at(1) : cards.at(3));

    if (round >= 1) {
        visibleCards.push_back(cards.at(4));
        visibleCards.push_back(cards.at(5));
        visibleCards.push_back(cards.at(6));
        if (round >= 2) {
            visibleCards.push_back(cards.at(7));
            if (round >= 3) {
                visibleCards.push_back(cards.at(8));
            }
        }
    }

    return visibleCards;
}

std::pair<int, GameState*> getCurrentNode(Template* schablone, std::vector<std::pair<char, float>> actionHistory) {

    // init with root
    int currentNode = *(schablone->structureList->worklist);
    GameState* currentGameState = new GameState();

    for (int i = 0; i < actionHistory.size(); i++) {

        pair<char, float> currentAction = actionHistory.at(i).first == 'r' ? pair<char, float>('r', getRaise(actionHistory.at(i).second)) : actionHistory.at(i);

        vector<pair<char, float>> possibleActions = currentGameState->getActions();

        std::vector<pair<char, float>>::iterator it = std::find(possibleActions.begin(), possibleActions.end(), currentAction);

        int index = it - possibleActions.begin();

        int currentNodeIndex = *(schablone->structureList->childrenWorklistPointers + currentNode) + index;

        currentNode = schablone->structureList->worklist[currentNodeIndex];

        currentGameState = currentGameState->handleAction(currentAction).gameState;

    }

    return pair<int, GameState*>(currentNode, currentGameState);

}